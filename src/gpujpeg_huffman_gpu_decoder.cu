#include "hip/hip_runtime.h"
/**
 * @file
 * Copyright (c) 2011-2020, CESNET z.s.p.o
 * Copyright (c) 2011, Silicon Genome, LLC.
 *
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "gpujpeg_huffman_gpu_decoder.h"
#include "gpujpeg_util.h"

/**
 * Entry of pre-built Huffman fast-decoding table.
 */
struct gpujpeg_table_huffman_decoder_entry {
    
    int value_nbits;
};

/** Number of code bits to be checked first (with high chance for the code to fit into this number of bits). */
#define QUICK_CHECK_BITS 10
#define QUICK_TABLE_ITEMS (4 * (1 << QUICK_CHECK_BITS))
// TODO: try to tweak QUICK table size and memory space

struct gpujpeg_huffman_gpu_decoder {
    /**
     * 4 pre-built tables for faster Huffman decoding (codewords up-to 16 bit length):
     *   - 0x00000 to 0x0ffff: luminance DC table
     *   - 0x10000 to 0x1ffff: luminance AC table
     *   - 0x20000 to 0x2ffff: chrominance DC table
     *   - 0x30000 to 0x3ffff: chrominance AC table
     *
     * Each entry consists of:
     *   - Number of bits of code corresponding to this entry (0 - 16, both inclusive) - bits 4 to 8
     *   - Number of run-length coded zeros before currently decoded coefficient + 1 (1 - 64, both inclusive) - bits 9 to 15
     *   - Number of bits representing the value of currently decoded coefficient (0 - 15, both inclusive) - bits 0 to 3
     * @code
     * bit #:    15                      9   8               4   3           0
     *         +---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+
     * value:  |      RLE zero count       |   code bit size   | value bit size|
     *         +---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+
     * @endcode
     */
    uint16_t *d_tables_full;

    /** Table with same format as the full table, except that all-zero-entry means that the full table should be consulted. */
    uint16_t *d_tables_quick;

    /** Natural order */
    int *d_order_natural;
};

#ifdef HUFFMAN_GPU_CONST_TABLES
/** Same table as above, but copied into constant memory */
__constant__ uint16_t gpujpeg_huffman_gpu_decoder_tables_quick_const[QUICK_TABLE_ITEMS];
 
/** Natural order in constant memory */
__constant__ int gpujpeg_huffman_gpu_decoder_order_natural[GPUJPEG_ORDER_NATURAL_SIZE];
#endif

// /**
//  * Fill more bit to current get buffer
//  * 
//  * @param get_bits
//  * @param get_buff
//  * @param data
//  * @param data_size
//  * @return void
//  */
// __device__ inline void
// gpujpeg_huffman_gpu_decoder_decode_fill_bit_buffer(int & get_bits, int & get_buff, uint8_t* & data, int & data_size)
// {
//     while ( get_bits < 25 ) {
//         //Are there some data?
//         if( data_size > 0 ) { 
//             // Attempt to read a byte
//             //printf("read byte %X 0x%X\n", (int)data, (unsigned char)*data);
//             unsigned char uc = *data++;
//             data_size--;            
// 
//             // If it's 0xFF, check and discard stuffed zero byte
//             if ( uc == 0xFF ) {
//                 while ( uc == 0xFF ) {
//                     //printf("read byte %X 0x%X\n", (int)data, (unsigned char)*data);
//                     uc = *data++;
//                     data_size--;
//                 }
// 
//                 if ( uc == 0 ) {
//                     // Found FF/00, which represents an FF data byte
//                     uc = 0xFF;
//                 } else {                
//                     // There should be enough bits still left in the data segment;
//                     // if so, just break out of the outer while loop.
//                     //if (m_nGetBits >= nbits)
//                     if ( get_bits >= 0 )
//                         break;
//                 }
//             }
// 
//             get_buff = (get_buff << 8) | ((int) uc);
//             get_bits += 8;            
//         }
//         else
//             break;
//     }
// }

/**
 * Loads at least specified number of bits into the register
 */
__device__ inline void
gpujpeg_huffman_gpu_decoder_load_bits(
                const unsigned int required_bit_count, unsigned int & r_bit,
                unsigned int & r_bit_count, uint4 * const s_byte, unsigned int & s_byte_idx
) {
    // Add bytes until have enough
    while(r_bit_count < required_bit_count) {
        // Load byte value and posibly skip next stuffed byte if loaded byte's value is 0xFF
        const uint8_t byte_value = ((const uint8_t*)s_byte)[s_byte_idx++];
        if((uint8_t)0xFF == byte_value) {
            s_byte_idx++;
        }
        
        // Add newly loaded byte to the buffer, updating bit count
        r_bit = (r_bit << 8) + byte_value;
        r_bit_count += 8;
    }
}


/**
 * Get bits
 * 
 * @param nbits  Number of bits to get
 * @param get_bits
 * @param get_buff
 * @param data
 * @param data_size
 * @return bits
 */
__device__ inline unsigned int
gpujpeg_huffman_gpu_decoder_get_bits(
                const unsigned int nbits, unsigned int & r_bit, unsigned int & r_bit_count, 
                uint4 * const s_byte, unsigned int & s_byte_idx)
{
    // load bits into the register if haven't got enough
    gpujpeg_huffman_gpu_decoder_load_bits(nbits, r_bit, r_bit_count, s_byte, s_byte_idx);
    
    // update remaining bit count
    r_bit_count -= nbits;
    
    // return bits 
    return (r_bit >> r_bit_count) & ((1 << nbits) - 1);
}

/**
 * Gets bits without removing them from the buffer.
 */
__device__ inline unsigned int
gpujpeg_huffman_gpu_decoder_peek_bits(
                const unsigned int nbits, unsigned int & r_bit, unsigned int & r_bit_count,
                uint4 * const s_byte, unsigned int & s_byte_idx)
{
    // load bits into the register if haven't got enough
    gpujpeg_huffman_gpu_decoder_load_bits(nbits, r_bit, r_bit_count, s_byte, s_byte_idx);
    
    // return bits 
    return (r_bit >> (r_bit_count - nbits)) & ((1 << nbits) - 1);
}

/**
 * Removes some bits from the buffer (assumes that they are there)
 */
__device__ inline void
gpujpeg_huffman_gpu_decoder_discard_bits(const unsigned int nb, unsigned int, unsigned int & r_bit_count) {
    r_bit_count -= nb;
}

/**
 * Special Huffman decode:
 * (1) For codes with length > 8
 * (2) For codes with length < 8 while data is finished
 * 
 * @param table
 * @param min_bits
 * @param get_bits
 * @param get_buff
 * @param data
 * @param data_size
 * @return int
 */
__device__ inline int
gpujpeg_huffman_gpu_decoder_decode_special_decode(
                const struct gpujpeg_table_huffman_decoder* const table, int min_bits, unsigned int & r_bit,
                unsigned int & r_bit_count, uint4 * const s_byte, unsigned int & s_byte_idx)
{
    // HUFF_DECODE has determined that the code is at least min_bits
    // bits long, so fetch that many bits in one swoop.
    int code = gpujpeg_huffman_gpu_decoder_get_bits(min_bits, r_bit, r_bit_count, s_byte, s_byte_idx);

    // Collect the rest of the Huffman code one bit at a time.
    // This is per Figure F.16 in the JPEG spec.
    int l = min_bits;
    while ( code > table->maxcode[l] ) {
        code <<= 1;
        code |= gpujpeg_huffman_gpu_decoder_get_bits(1, r_bit, r_bit_count, s_byte, s_byte_idx);
        l++;
    }

    // With garbage input we may reach the sentinel value l = 17.
    if ( l > 16 ) {
        // Fake a zero as the safest result
        return 0;
    }
    
    return table->huffval[table->valptr[l] + (int)(code - table->mincode[l])];
}

/**
 * To find dc or ac value according to code and its bit length s
 */
__device__ inline int
gpujpeg_huffman_gpu_decoder_value_from_category(int nbits, int code)
{
    // TODO: try to replace with __constant__ table lookup
    return code < ((1 << nbits) >> 1) ? (code + ((-1) << nbits) + 1) : code;
    
//     // Method 1: 
//     // On some machines, a shift and add will be faster than a table lookup.
//     // #define HUFF_EXTEND(x,s) ((x)< (1<<((s)-1)) ? (x) + (((-1)<<(s)) + 1) : (x))
// 
//     // Method 2: Table lookup
//     // If (offset < half[category]), then value is below zero
//     // Otherwise, value is above zero, and just the offset 
//     // entry n is 2**(n-1)
//     const int half[16] =    { 
//         0x0000, 0x0001, 0x0002, 0x0004, 0x0008, 0x0010, 0x0020, 0x0040, 
//         0x0080, 0x0100, 0x0200, 0x0400, 0x0800, 0x1000, 0x2000, 0x4000
//     };
// 
//     //start[i] is the starting value in this category; surely it is below zero
//     // entry n is (-1 << n) + 1
//     const int start[16] = { 
//         0, ((-1)<<1) + 1, ((-1)<<2) + 1, ((-1)<<3) + 1, ((-1)<<4) + 1,
//         ((-1)<<5) + 1, ((-1)<<6) + 1, ((-1)<<7) + 1, ((-1)<<8) + 1,
//         ((-1)<<9) + 1, ((-1)<<10) + 1, ((-1)<<11) + 1, ((-1)<<12) + 1,
//         ((-1)<<13) + 1, ((-1)<<14) + 1, ((-1)<<15) + 1 
//     };    
// 
//     return (code < half[nbits]) ? (code + start[nbits]) : code;    
}

/**
 * Decodes next coefficient, updating its output index
 * 
 * @param table
 * @param get_bits
 * @param get_buff
 * @param data
 * @param data_size
 * @return int
 */
__device__ inline int
gpujpeg_huffman_gpu_decoder_get_coefficient(struct gpujpeg_huffman_gpu_decoder huffman_gpu_decoder,
                unsigned int & r_bit, unsigned int & r_bit_count, uint4* const s_byte,
                unsigned int & s_byte_idx, const unsigned int table_offset, unsigned int & coefficient_idx)
{
    // Peek next 16 bits and use them as an index into decoder table to find all the info.
    const unsigned int table_idx = table_offset + gpujpeg_huffman_gpu_decoder_peek_bits(16, r_bit, r_bit_count, s_byte, s_byte_idx);
    
    // Try the quick table first (use the full table only if not succeded with the quick table)
#ifdef HUFFMAN_GPU_CONST_TABLES
    unsigned int packed_info = gpujpeg_huffman_gpu_decoder_tables_quick_const[table_idx >> (16 - QUICK_CHECK_BITS)];
#else
    unsigned int packed_info = huffman_gpu_decoder.d_tables_quick[table_idx >> (16 - QUICK_CHECK_BITS)];
#endif
    if(0 == packed_info) {
        packed_info = huffman_gpu_decoder.d_tables_full[table_idx];
    }
    
    // remove the right number of bits from the bit buffer
    gpujpeg_huffman_gpu_decoder_discard_bits((packed_info >> 4) & 0x1F, r_bit, r_bit_count);
    
    // update coefficient index by skipping run-length encoded zeros
    coefficient_idx += packed_info >> 9;
    
    // read coefficient bits and decode the coefficient from them
    const unsigned int value_nbits = packed_info & 0xF;
    const unsigned int value_code = gpujpeg_huffman_gpu_decoder_get_bits(value_nbits, r_bit, r_bit_count, s_byte, s_byte_idx);
    
    // return deocded coefficient
    return gpujpeg_huffman_gpu_decoder_value_from_category(value_nbits, value_code);
}

/**
 * Decode one 8x8 block
 *
 * @return 0 if succeeds, otherwise nonzero
 */
__device__ inline int
gpujpeg_huffman_gpu_decoder_decode_block(
    struct gpujpeg_huffman_gpu_decoder huffman_gpu_decoder,
    int & dc, int16_t* const data_output, const unsigned int dc_table_offset, const unsigned int ac_table_offset,
    unsigned int & r_bit, unsigned int & r_bit_count, uint4* const s_byte,
    unsigned int & s_byte_idx, const uint4* & d_byte, unsigned int & d_byte_chunk_count)
{
    // TODO: try unified decoding of DC/AC coefficients
    
    // Index of next coefficient to be decoded (in ZIG-ZAG order)
    unsigned int coefficient_idx = 0;
    
    // Section F.2.2.1: decode the DC coefficient difference
    // Get the coefficient value (using DC coding table)
    int dc_coefficient_value = gpujpeg_huffman_gpu_decoder_get_coefficient(huffman_gpu_decoder, r_bit, r_bit_count, s_byte, s_byte_idx, dc_table_offset, coefficient_idx);

    // Convert DC difference to actual value, update last_dc_val
    dc = dc_coefficient_value += dc;

    // Output the DC coefficient (assumes gpujpeg_natural_order[0] = 0)
    // TODO: try to skip saving of zero coefficients
    data_output[0] = dc_coefficient_value;
    
    // TODO: error check: coefficient_idx must still be 0 in valid codestreams
    coefficient_idx = 1;
    
    // Section F.2.2.2: decode the AC coefficients
    // Since zeroes are skipped, output area must be cleared beforehand
    do {
        // Possibly load more bytes into shared buffer from global memory
        if(s_byte_idx >= 16) {
            // Move remaining bytes to begin and update index of next byte
            s_byte[0] = s_byte[1];
            s_byte_idx -= 16;
            
            // Load another byte chunk from global memory only if there is one
            if(d_byte_chunk_count) {
                s_byte[1] = *(d_byte++);
                d_byte_chunk_count--;
            }
        }
        
        // decode next coefficient, updating its destination index
        const int coefficient_value = gpujpeg_huffman_gpu_decoder_get_coefficient(huffman_gpu_decoder, r_bit, r_bit_count, s_byte, s_byte_idx, ac_table_offset, coefficient_idx);
        
        // stop with this block if have all coefficients
        if(coefficient_idx > 64) {
            break;
        }
        
        // save the coefficient   TODO: try to ommit saving 0 coefficients
#ifdef HUFFMAN_GPU_CONST_TABLES
        data_output[gpujpeg_huffman_gpu_decoder_order_natural[coefficient_idx - 1]] = coefficient_value;
#else
        data_output[huffman_gpu_decoder.d_order_natural[coefficient_idx - 1]] = coefficient_value;
#endif
    } while(coefficient_idx < 64);
    
    return 0;
}


/**
 * Huffman decoder kernel
 * 
 * @return void
 */
template <bool SINGLE_COMP, int THREADS_PER_TBLOCK>
__global__ void
#if __CUDA_ARCH__ < 200
__launch_bounds__(THREADS_PER_TBLOCK, 2)
#else
__launch_bounds__(THREADS_PER_TBLOCK, 6)
#endif
gpujpeg_huffman_decoder_decode_kernel(
    struct gpujpeg_huffman_gpu_decoder huffman_gpu_decoder,
    struct gpujpeg_component* d_component,
    struct gpujpeg_segment* d_segment,
    int comp_count,
    int segment_count, 
    uint8_t* d_data_compressed,
    const uint64_t* d_block_list,
    int16_t* d_data_quantized
) {
    int segment_index = blockIdx.x * THREADS_PER_TBLOCK + threadIdx.x;
    if ( segment_index >= segment_count )
        return;
    
    struct gpujpeg_segment* segment = &d_segment[segment_index];
    
    // Byte buffers in shared memory
    __shared__ uint4 s_byte_all[2 * THREADS_PER_TBLOCK]; // 32 bytes per thread
    uint4 * const s_byte = s_byte_all + 2 * threadIdx.x;
    
    // Last DC coefficient values   TODO: try to move into shared memory
    int dc[GPUJPEG_MAX_COMPONENT_COUNT];
    for ( int comp = 0; comp < GPUJPEG_MAX_COMPONENT_COUNT; comp++ )
        dc[comp] = 0;
        
    // Get aligned compressed data chunk pointer and load first 2 chunks of the data
    const unsigned int d_byte_begin_idx = segment->data_compressed_index;
    const unsigned int d_byte_begin_idx_aligned = d_byte_begin_idx & ~15; // loading 16byte chunks
    const uint4* d_byte = (uint4*)(d_data_compressed + d_byte_begin_idx_aligned);
    
    // Get number of remaining global memory byte chunks (not to read bytes out of buffer)
    const unsigned int d_byte_end_idx_aligned = (d_byte_begin_idx + segment->data_compressed_size + 15) & ~15;
    unsigned int d_byte_chunk_count = (d_byte_end_idx_aligned - d_byte_begin_idx_aligned) / 16;
    
    // Load first 2 chunks of compressed data into the shared memory buffer and remember index of first code byte (skipping bytes read due to alignment)
    s_byte[0] = d_byte[0];
    s_byte[1] = d_byte[1];
    d_byte += 2;
    d_byte_chunk_count = max(d_byte_chunk_count, 2) - 2;
    unsigned int s_byte_idx = d_byte_begin_idx - d_byte_begin_idx_aligned;
    
    // bits loaded into the register and their count
    unsigned int r_bit_count = 0;
    unsigned int r_bit = 0; // LSB-aligned
    
    // Non-interleaving mode
    if ( SINGLE_COMP ) {
        // Get component for current scan
        const struct gpujpeg_component* const component = d_component + segment->scan_index; 
        
        // Get huffman tables offset
        const unsigned int dc_table_offset = component->dc_huff_idx * 0x20000;
        const unsigned int ac_table_offset = component->ac_huff_idx * 0x20000 + 0x10000;

        // Size of MCUs in this segment's component
        const int component_mcu_size = component->mcu_size;
        
        // Pointer to first MCU's output block
        int16_t* block = component->d_data_quantized + segment->scan_segment_index * component->segment_mcu_count * component_mcu_size;
        
        // Encode MCUs in segment
        for ( int mcu_index = 0; mcu_index < segment->mcu_count; mcu_index++ ) {
            // Encode 8x8 block
            if ( gpujpeg_huffman_gpu_decoder_decode_block(huffman_gpu_decoder, dc[0], block, dc_table_offset, ac_table_offset, r_bit, r_bit_count, s_byte, s_byte_idx, d_byte, d_byte_chunk_count) != 0 )
                break;
            
            // advance to next block
            block += component_mcu_size;
        } 
    }
    // Interleaving mode
    else {
        // Pointer to segment's list of 8x8 blocks and their count
        const uint64_t* packed_block_info_ptr = d_block_list + segment->block_index_list_begin;
        
        // Encode all blocks
        for(int block_count = segment->block_count; block_count--;) {
            // Get pointer to next block input data and info about its color type
            const uint64_t packed_block_info = *(packed_block_info_ptr++);
            
            // Get coder parameters
            const int last_dc_idx = packed_block_info & 0x7f;
            
            // Get offset to right part of huffman table
            const unsigned int dc_huffman_table_offset = d_component[last_dc_idx].dc_huff_idx * 0x20000;
            const unsigned int ac_huffman_table_offset = d_component[last_dc_idx].ac_huff_idx * 0x20000 + 0x10000;
            
            // Source data pointer
            int16_t* block = d_data_quantized + (packed_block_info >> 8);
            
            // Encode 8x8 block
            gpujpeg_huffman_gpu_decoder_decode_block(huffman_gpu_decoder, dc[last_dc_idx], block, dc_huffman_table_offset, ac_huffman_table_offset, r_bit, r_bit_count, s_byte, s_byte_idx, d_byte, d_byte_chunk_count);
        }
        
        
//         // Encode MCUs in segment
//         for ( int mcu_index = 0; mcu_index < segment->mcu_count; mcu_index++ ) {
//             
//             
//             
//             
//             
//             
//             
//             
//             //assert(segment->scan_index == 0);
//             for ( int comp = 0; comp < comp_count; comp++ ) {
//                 struct gpujpeg_component* component = &d_component[comp];
// 
//                 // Prepare mcu indexes
//                 int mcu_index_x = (segment_index * component->segment_mcu_count + mcu_index) % component->mcu_count_x;
//                 int mcu_index_y = (segment_index * component->segment_mcu_count + mcu_index) / component->mcu_count_x;
//                 // Compute base data index
//                 int data_index_base = mcu_index_y * (component->mcu_size * component->mcu_count_x) + mcu_index_x * (component->mcu_size_x * GPUJPEG_BLOCK_SIZE);
//                 
//                 // For all vertical 8x8 blocks
//                 for ( int y = 0; y < component->sampling_factor.vertical; y++ ) {
//                     // Compute base row data index
//                     int data_index_row = data_index_base + y * (component->mcu_count_x * component->mcu_size_x * GPUJPEG_BLOCK_SIZE);
//                     // For all horizontal 8x8 blocks
//                     for ( int x = 0; x < component->sampling_factor.horizontal; x++ ) {
//                         // Compute 8x8 block data index
//                         int data_index = data_index_row + x * GPUJPEG_BLOCK_SIZE * GPUJPEG_BLOCK_SIZE;
//                         
//                         // Get component data for MCU
//                         int16_t* block = &component->d_data_quantized[data_index];
//                         
//                         // Get coder parameters
//                         int & component_dc = dc[comp];
//             
//                         // Get huffman tables offset
//                         const unsigned int table_offset = component->type == GPUJPEG_COMPONENT_LUMINANCE ? 0x00000 : 0x20000;
//                         
//                         // Encode 8x8 block
//                         gpujpeg_huffman_gpu_decoder_decode_block(component_dc, block, table_offset, r_bit, r_bit_count, s_byte, s_byte_idx, d_byte, d_byte_chunk_count);
//                     }
//                 }
//             }
//         }
    }
}

/**
 * Setup of one Huffman table entry for fast decoding.
 * @param bits  bits to extract one codeword from (first bit is bit #15, then #14, ... last is #0)
 * @param d_table_src  source (slow-decoding) table pointer
 * @param d_table_dest  destination (fast-decoding) table pointer
 */
__device__ void
gpujpeg_huffman_gpu_decoder_table_setup(
    struct gpujpeg_huffman_gpu_decoder huffman_gpu_decoder,
    const int bits, 
    const struct gpujpeg_table_huffman_decoder* const d_table_src,
    const int table_idx
) {
    // Decode one codeword from given bits to get following:
    //  - minimal number of bits actually needed to decode the codeword (up to 16 bits, 0 for invalid ones)
    //  - category ID represented by the codeword, consisting from:
    //      - number of run-length-coded preceding zeros (up to 16, or 63 for both special end-of block symbol or invalid codewords)
    //      - bit-size of the actual value of coefficient (up to 16, 0 for invalid ones)
    int code_nbits = 1, category_id = 0;
    
    // First, decode codeword length (This is per Figure F.16 in the JPEG spec.)
    int code_value = bits >> 15; // only single bit initially
    while ( code_value > d_table_src->maxcode[code_nbits] ) {
        code_value = bits >> (16 - ++code_nbits); // not enough to decide => try more bits
    }
    
    // With garbage input we may reach the sentinel value l = 17.
    if ( code_nbits > 16 ) {
        code_nbits = 0;
        // category ID remains 0 for invalid symbols from garbage input
    } else {
        category_id = d_table_src->huffval[d_table_src->valptr[code_nbits] + code_value - d_table_src->mincode[code_nbits]];
    }
    
    // decompose category number into 1 + number of run-length coded zeros and length of the value
    // (special category #0 contains all invalid codes and special end-of-block code -- all of those codes 
    // should terminate block decoding => use 64 run-length zeros and 0 value bits for such symbols)
    const int value_nbits = 0xF & category_id;
    const int rle_zero_count = category_id ? min(1 + (category_id >> 4), 64) : 64;
    
    // save all the info into the right place in the destination table
    const int packed_info = (rle_zero_count << 9) + (code_nbits << 4) + value_nbits;
    huffman_gpu_decoder.d_tables_full[(table_idx << 16) + bits] = packed_info;
    
    // some threads also save entries into the quick table
    const int dest_idx_quick = bits >> (16 - QUICK_CHECK_BITS);
    if(bits == (dest_idx_quick << (16 - QUICK_CHECK_BITS))) {
        // save info also into the quick table if number of required bits is less than quick 
        // check bit count, otherwise put 0 there to indicate that full table lookup consultation is needed
        huffman_gpu_decoder.d_tables_quick[(table_idx << QUICK_CHECK_BITS) + dest_idx_quick] = code_nbits <= QUICK_CHECK_BITS ? packed_info : 0;
    }
}

/**
 * Huffman decoder table setup kernel
 * (Based on the original table, this kernel prepares another table, which is more suitable for fast decoding.)
 */
__global__ void
gpujpeg_huffman_decoder_table_kernel(
                struct gpujpeg_huffman_gpu_decoder huffman_gpu_decoder,
                const struct gpujpeg_table_huffman_decoder* const d_table_y_dc,
                const struct gpujpeg_table_huffman_decoder* const d_table_y_ac,
                const struct gpujpeg_table_huffman_decoder* const d_table_cbcr_dc,
                const struct gpujpeg_table_huffman_decoder* const d_table_cbcr_ac
) {
    // Each thread uses all 4 Huffman tables to "decode" one symbol from its unique 16bits.
    const int idx = threadIdx.x + blockIdx.x * blockDim.x;
    gpujpeg_huffman_gpu_decoder_table_setup(huffman_gpu_decoder, idx, d_table_y_dc, 0);
    gpujpeg_huffman_gpu_decoder_table_setup(huffman_gpu_decoder, idx, d_table_y_ac, 1);
    gpujpeg_huffman_gpu_decoder_table_setup(huffman_gpu_decoder, idx, d_table_cbcr_dc, 2);
    gpujpeg_huffman_gpu_decoder_table_setup(huffman_gpu_decoder, idx, d_table_cbcr_ac, 3);
}

/* Documented at declaration */
struct gpujpeg_huffman_gpu_decoder *
gpujpeg_huffman_gpu_decoder_init()
{
    struct gpujpeg_huffman_gpu_decoder *huffman_gpu_decoder = (struct gpujpeg_huffman_gpu_decoder *) calloc(1, sizeof(struct gpujpeg_huffman_gpu_decoder));

#ifdef HUFFMAN_GPU_CONST_TABLES
    // Copy natural order to constant device memory
    hipMemcpyToSymbol(HIP_SYMBOL(
        gpujpeg_huffman_gpu_decoder_order_natural),
        gpujpeg_order_natural,
        GPUJPEG_ORDER_NATURAL_SIZE * sizeof(int),
        0,
        hipMemcpyHostToDevice
    );
    gpujpeg_cuda_check_error("Huffman decoder init", gpujpeg_huffman_gpu_decoder_destroy(huffman_gpu_decoder); return NULL);
#else
    hipMalloc((void**)&huffman_gpu_decoder->d_order_natural, GPUJPEG_ORDER_NATURAL_SIZE * sizeof(int));
    gpujpeg_cuda_check_error("Huffman GPU decoder natural order table allocation", gpujpeg_huffman_gpu_decoder_destroy(huffman_gpu_decoder); return NULL);
    hipMemcpy(
        huffman_gpu_decoder->d_order_natural,
        gpujpeg_order_natural,
        GPUJPEG_ORDER_NATURAL_SIZE * sizeof(int),
        hipMemcpyHostToDevice
    );
    gpujpeg_cuda_check_error("Huffman GPU decoder natural order table copy", gpujpeg_huffman_gpu_decoder_destroy(huffman_gpu_decoder); return NULL);
#endif

    hipMalloc((void**)&huffman_gpu_decoder->d_tables_full, 4 * (1 << 16) * sizeof(uint16_t));
    gpujpeg_cuda_check_error("Huffman GPU decoder full table allocation", gpujpeg_huffman_gpu_decoder_destroy(huffman_gpu_decoder); return NULL);

    hipMalloc((void**)&huffman_gpu_decoder->d_tables_quick, QUICK_TABLE_ITEMS * sizeof(uint16_t));
    gpujpeg_cuda_check_error("Huffman GPU decoder quick table allocation", gpujpeg_huffman_gpu_decoder_destroy(huffman_gpu_decoder); return NULL);
    
    return huffman_gpu_decoder;
}

void
gpujpeg_huffman_gpu_decoder_destroy(struct gpujpeg_huffman_gpu_decoder *huffman_gpu_decoder)
{
    if (huffman_gpu_decoder == NULL) {
        return;
    }

    hipFree(huffman_gpu_decoder->d_order_natural);
    hipFree(huffman_gpu_decoder->d_tables_full);
    hipFree(huffman_gpu_decoder->d_tables_quick);
    free(huffman_gpu_decoder);
}

/* Documented at declaration */
int
gpujpeg_huffman_gpu_decoder_decode(struct gpujpeg_decoder* decoder)
{    
    // Get coder
    struct gpujpeg_coder* coder = &decoder->coder;
    
    // assert(coder->param.restart_interval > 0);
    
    int comp_count = 1;
    if (coder->param.interleaved == 1) {
        comp_count = coder->param.comp_count;
    }
    assert(comp_count >= 1 && comp_count <= GPUJPEG_MAX_COMPONENT_COUNT);
    
    // Number of decoder kernel threads per each threadblock
    enum { THREADS_PER_TBLOCK = 192 };
    
    // Configure more Shared memory for both kernels
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpujpeg_huffman_decoder_table_kernel), hipFuncCachePreferShared);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpujpeg_huffman_decoder_decode_kernel<true), THREADS_PER_TBLOCK>, hipFuncCachePreferShared);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpujpeg_huffman_decoder_decode_kernel<false), THREADS_PER_TBLOCK>, hipFuncCachePreferShared);
    
    // Setup GPU tables (one thread for each of 65536 entries)
    gpujpeg_huffman_decoder_table_kernel<<<256, 256, 0, decoder->stream>>>(
        *decoder->huffman_gpu_decoder,
        decoder->d_table_huffman[GPUJPEG_COMPONENT_LUMINANCE][GPUJPEG_HUFFMAN_DC],
        decoder->d_table_huffman[GPUJPEG_COMPONENT_LUMINANCE][GPUJPEG_HUFFMAN_AC],
        decoder->d_table_huffman[GPUJPEG_COMPONENT_CHROMINANCE][GPUJPEG_HUFFMAN_DC],
        decoder->d_table_huffman[GPUJPEG_COMPONENT_CHROMINANCE][GPUJPEG_HUFFMAN_AC]
    );
    gpujpeg_cuda_check_error("Huffman decoder table setup failed", return -1);

#ifdef HUFFMAN_GPU_CONST_TABLES
    // Copy quick decoding table into constant memory
    hipMemcpyToSymbolAsync(HIP_SYMBOL(
        gpujpeg_huffman_gpu_decoder_tables_quick_const),
        decoder->huffman_gpu_decoder->d_tables_quick,
        sizeof(*decoder->huffman_gpu_decoder->d_tables_quick) * QUICK_TABLE_ITEMS,
        0,
        hipMemcpyDeviceToDevice,
        decoder->stream
    );
    gpujpeg_cuda_check_error("Huffman decoder table copy failed", return -1);
#endif

    for (int comp = 0; comp < coder->param.comp_count; comp++) {
        coder->component[comp].dc_huff_idx = decoder->comp_table_huffman_map[comp][GPUJPEG_HUFFMAN_DC];
        coder->component[comp].ac_huff_idx = decoder->comp_table_huffman_map[comp][GPUJPEG_HUFFMAN_AC];
    }
    // Copy updated components to device memory
    hipMemcpyAsync(coder->d_component, coder->component, coder->param.comp_count * sizeof(struct gpujpeg_component), hipMemcpyHostToDevice, decoder->stream);
    gpujpeg_cuda_check_error("Coder component copy", return 0);
    
    // Run decoding kernel
    dim3 thread(THREADS_PER_TBLOCK);
    dim3 grid(gpujpeg_div_and_round_up(decoder->segment_count, THREADS_PER_TBLOCK));
    if(comp_count == 1) {
        gpujpeg_huffman_decoder_decode_kernel<true, THREADS_PER_TBLOCK><<<grid, thread, 0, decoder->stream>>>(
            *decoder->huffman_gpu_decoder,
            coder->d_component, 
            coder->d_segment, 
            comp_count,
            decoder->segment_count,
            coder->d_data_compressed,
            coder->d_block_list,
            coder->d_data_quantized
        );
    } else {
        gpujpeg_huffman_decoder_decode_kernel<false, THREADS_PER_TBLOCK><<<grid, thread, 0, decoder->stream>>>(
            *decoder->huffman_gpu_decoder,
            coder->d_component, 
            coder->d_segment, 
            comp_count,
            decoder->segment_count,
            coder->d_data_compressed,
            coder->d_block_list,
            coder->d_data_quantized
        );
    }
    gpujpeg_cuda_check_error("Huffman decoding failed", return -1);
    
    return 0;
}

/* vi: set expandtab sw=4 : */
