#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011-2021, CESNET z.s.p.o
 * Copyright (c) 2011, Silicon Genome, LLC.
 *
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */
/**
 * @file
 * @brief
 * This file contains postprocessors a common format for computational kernels
 * to raw image. It also does color space transformations.
 */

#include "gpujpeg_colorspace.h"
#include "gpujpeg_preprocessor_common.h"
#include "gpujpeg_postprocessor.h"
#include "gpujpeg_util.h"

/**
 * Store value to component data buffer in specified position by buffer size and subsampling
 *
 * @param value
 * @param position_x
 * @param position_y
 * @param comp
 */
template<
    uint8_t s_samp_factor_h = GPUJPEG_DYNAMIC,
    uint8_t s_samp_factor_v = GPUJPEG_DYNAMIC
>
struct gpujpeg_preprocessor_comp_to_raw_load
{
    static __device__ void
    perform(uint8_t & value, int position_x, int position_y, struct gpujpeg_preprocessor_data_component & comp)
    {
        uint8_t samp_factor_h = s_samp_factor_h;
        if ( samp_factor_h == GPUJPEG_DYNAMIC ) {
            samp_factor_h = comp.sampling_factor.horizontal;
        }
        uint8_t samp_factor_v = s_samp_factor_v;
        if ( samp_factor_v == GPUJPEG_DYNAMIC ) {
            samp_factor_v = comp.sampling_factor.vertical;
        }

        position_x = position_x / samp_factor_h;
        position_y = position_y / samp_factor_v;

        int data_position = position_y * comp.data_width + position_x;
        value = comp.d_data[data_position];
    }
};
template<>
struct gpujpeg_preprocessor_comp_to_raw_load<1, 1>
{
    static __device__ void
    perform(uint8_t & value, int position_x, int position_y, struct gpujpeg_preprocessor_data_component & comp)
    {
        int data_position = position_y * comp.data_width + position_x;
        value = comp.d_data[data_position];
    }
};

template<enum gpujpeg_pixel_format pixel_format>
inline __device__ void gpujpeg_comp_to_raw_store(uint8_t *d_data_raw, int &image_width, int &image_height, int &image_position, uint8_t &r1, uint8_t &r2, uint8_t &r3);

template<>
inline __device__ void gpujpeg_comp_to_raw_store<GPUJPEG_U8>(uint8_t *d_data_raw, int &image_width, int &image_height, int &image_position, uint8_t &r1, uint8_t &r2, uint8_t &r3)
{
    d_data_raw[image_position] = r1;
}

template<>
inline __device__ void gpujpeg_comp_to_raw_store<GPUJPEG_444_U8_P012>(uint8_t *d_data_raw, int &image_width, int &image_height, int &image_position, uint8_t &r1, uint8_t &r2, uint8_t &r3)
{
    image_position = image_position * 3;
    d_data_raw[image_position + 0] = r1;
    d_data_raw[image_position + 1] = r2;
    d_data_raw[image_position + 2] = r3;
}

template<>
inline __device__ void gpujpeg_comp_to_raw_store<GPUJPEG_444_U8_P012A>(uint8_t *d_data_raw, int &image_width, int &image_height, int &image_position, uint8_t &r1, uint8_t &r2, uint8_t &r3)
{
    image_position = image_position * 4;
    d_data_raw[image_position + 0] = r1;
    d_data_raw[image_position + 1] = r2;
    d_data_raw[image_position + 2] = r3;
    d_data_raw[image_position + 3] = 0xFF;
}

template<>
inline __device__ void gpujpeg_comp_to_raw_store<GPUJPEG_444_U8_P012Z>(uint8_t *d_data_raw, int &image_width, int &image_height, int &image_position, uint8_t &r1, uint8_t &r2, uint8_t &r3)
{
    image_position = image_position * 4;
    d_data_raw[image_position + 0] = r1;
    d_data_raw[image_position + 1] = r2;
    d_data_raw[image_position + 2] = r3;
    d_data_raw[image_position + 3] = 0x0;
}

template<>
inline __device__ void gpujpeg_comp_to_raw_store<GPUJPEG_444_U8_P0P1P2>(uint8_t *d_data_raw, int &image_width, int &image_height, int &image_position, uint8_t &r1, uint8_t &r2, uint8_t &r3)
{
    d_data_raw[image_position] = r1;
    d_data_raw[image_width * image_height + image_position] = r2;
    d_data_raw[2 * image_width * image_height + image_position] = r3;
}

template<>
inline __device__ void gpujpeg_comp_to_raw_store<GPUJPEG_422_U8_P0P1P2>(uint8_t *d_data_raw, int &image_width, int &image_height, int &image_position, uint8_t &r1, uint8_t &r2, uint8_t &r3)
{
    d_data_raw[image_position] = r1;
    if ( (image_position % 2) == 0 ) {
        d_data_raw[image_width * image_height + image_position / 2] = r2;
        d_data_raw[image_width * image_height + image_height * ((image_width + 1) / 2) + image_position / 2] = r3;
    }
}

template<>
inline __device__ void gpujpeg_comp_to_raw_store<GPUJPEG_422_U8_P1020>(uint8_t *d_data_raw, int &image_width, int &image_height, int &image_position, uint8_t &r1, uint8_t &r2, uint8_t &r3)
{
    image_position = image_position * 2;
    d_data_raw[image_position + 1] = r1;
    if ( (image_position % 2) == 0 )
        d_data_raw[image_position + 0] = r2;
    else
        d_data_raw[image_position + 0] = r3;
}

/**
 * Kernel - Copy three separated component buffers into target image data
 *
 * @param d_c1  First component buffer
 * @param d_c2  Second component buffer
 * @param d_c3  Third component buffer
 * @param d_target  Image target data
 * @param pixel_count  Number of pixels to copy
 * @return void
 */
typedef void (*gpujpeg_preprocessor_decode_kernel)(struct gpujpeg_preprocessor_data data, uint8_t* d_data_raw, int image_width, int image_height);

template<
    enum gpujpeg_color_space color_space_internal,
    enum gpujpeg_color_space color_space,
    enum gpujpeg_pixel_format pixel_format,
    uint8_t s_comp1_samp_factor_h, uint8_t s_comp1_samp_factor_v,
    uint8_t s_comp2_samp_factor_h, uint8_t s_comp2_samp_factor_v,
    uint8_t s_comp3_samp_factor_h, uint8_t s_comp3_samp_factor_v
>
__global__ void
gpujpeg_preprocessor_comp_to_raw_kernel(struct gpujpeg_preprocessor_data data, uint8_t* d_data_raw, int image_width, int image_height)
{
    int x  = threadIdx.x;
    int gX = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
    int image_position = gX + x;
    if ( image_position >= (image_width * image_height) )
        return;
    int image_position_x = image_position % image_width;
    int image_position_y = image_position / image_width;

    // Load
    uint8_t r1;
    uint8_t r2;
    uint8_t r3;
    gpujpeg_preprocessor_comp_to_raw_load<s_comp1_samp_factor_h, s_comp1_samp_factor_v>::perform(r1, image_position_x, image_position_y, data.comp[0]);
    gpujpeg_preprocessor_comp_to_raw_load<s_comp2_samp_factor_h, s_comp2_samp_factor_v>::perform(r2, image_position_x, image_position_y, data.comp[1]);
    gpujpeg_preprocessor_comp_to_raw_load<s_comp3_samp_factor_h, s_comp3_samp_factor_v>::perform(r3, image_position_x, image_position_y, data.comp[2]);

    // Color transform
    gpujpeg_color_transform<color_space_internal, color_space>::perform(r1, r2, r3);

    // Save
    gpujpeg_comp_to_raw_store<pixel_format>(d_data_raw, image_width, image_height, image_position, r1, r2, r3);

}

/**
 * Select preprocessor decode kernel
 *
 * @param decoder
 * @return kernel
 */
template<enum gpujpeg_color_space color_space_internal>
gpujpeg_preprocessor_decode_kernel
gpujpeg_preprocessor_select_decode_kernel(struct gpujpeg_coder* coder)
{
    gpujpeg_preprocessor_sampling_factor_t sampling_factor = gpujpeg_preprocessor_make_sampling_factor(
        coder->sampling_factor.horizontal / coder->component[0].sampling_factor.horizontal,
        coder->sampling_factor.vertical / coder->component[0].sampling_factor.vertical,
        coder->sampling_factor.horizontal / coder->component[1].sampling_factor.horizontal,
        coder->sampling_factor.vertical / coder->component[1].sampling_factor.vertical,
        coder->sampling_factor.horizontal / coder->component[2].sampling_factor.horizontal,
        coder->sampling_factor.vertical / coder->component[2].sampling_factor.vertical
    );

#define RETURN_KERNEL_IF(PIXEL_FORMAT, COLOR, P1, P2, P3, P4, P5, P6) \
    if ( sampling_factor == gpujpeg_preprocessor_make_sampling_factor(P1, P2, P3, P4, P5, P6) ) { \
        int max_h = max(P1, max(P3, P5)); \
        int max_v = max(P2, max(P4, P6)); \
        if ( coder->param.verbose >= 1 ) { \
            printf("Using faster kernel for postprocessor (precompiled %dx%d, %dx%d, %dx%d).\n", max_h / P1, max_v / P2, max_h / P3, max_v / P4, max_h / P5, max_v / P6); \
        } \
        if ( PIXEL_FORMAT == GPUJPEG_U8 ) { \
            return &gpujpeg_preprocessor_comp_to_raw_kernel<color_space_internal, COLOR, GPUJPEG_U8, P1, P2, P3, P4, P5, P6>; \
        } else if ( PIXEL_FORMAT == GPUJPEG_444_U8_P012 ) { \
            return &gpujpeg_preprocessor_comp_to_raw_kernel<color_space_internal, COLOR, GPUJPEG_444_U8_P012, P1, P2, P3, P4, P5, P6>; \
        } else if ( PIXEL_FORMAT == GPUJPEG_444_U8_P012A ) { \
            return &gpujpeg_preprocessor_comp_to_raw_kernel<color_space_internal, COLOR, GPUJPEG_444_U8_P012A, P1, P2, P3, P4, P5, P6>; \
        } else if ( PIXEL_FORMAT == GPUJPEG_444_U8_P012Z ) { \
            return &gpujpeg_preprocessor_comp_to_raw_kernel<color_space_internal, COLOR, GPUJPEG_444_U8_P012Z, P1, P2, P3, P4, P5, P6>; \
        } else if ( coder->param_image.pixel_format == GPUJPEG_422_U8_P1020 ) { \
            return &gpujpeg_preprocessor_comp_to_raw_kernel<color_space_internal, COLOR, GPUJPEG_422_U8_P1020, P1, P2, P3, P4, P5, P6>; \
        } else if ( coder->param_image.pixel_format == GPUJPEG_444_U8_P0P1P2 ) { \
            return &gpujpeg_preprocessor_comp_to_raw_kernel<color_space_internal, COLOR, GPUJPEG_444_U8_P0P1P2, P1, P2, P3, P4, P5, P6>; \
        } else if ( coder->param_image.pixel_format == GPUJPEG_422_U8_P0P1P2 ) { \
            return &gpujpeg_preprocessor_comp_to_raw_kernel<color_space_internal, COLOR, GPUJPEG_422_U8_P0P1P2, P1, P2, P3, P4, P5, P6>; \
        } else { \
            assert(false); \
        } \
    }
#define RETURN_KERNEL(PIXEL_FORMAT, COLOR) \
    RETURN_KERNEL_IF(PIXEL_FORMAT, COLOR, 1, 1, 1, 1, 1, 1) \
    else RETURN_KERNEL_IF(PIXEL_FORMAT, COLOR, 1, 1, 2, 2, 2, 2) \
    else RETURN_KERNEL_IF(PIXEL_FORMAT, COLOR, 1, 1, 1, 2, 1, 2) \
    else RETURN_KERNEL_IF(PIXEL_FORMAT, COLOR, 1, 1, 2, 1, 2, 1) \
    else RETURN_KERNEL_IF(PIXEL_FORMAT, COLOR, 1, 1, 4, 4, 4, 4) \
    else { \
        if ( coder->param.verbose >= 1 ) { \
            printf("Using slower kernel for postprocessor (dynamic %dx%d, %dx%d, %dx%d).\n", coder->component[0].sampling_factor.horizontal, coder->component[0].sampling_factor.vertical, coder->component[1].sampling_factor.horizontal, coder->component[1].sampling_factor.vertical, coder->component[2].sampling_factor.horizontal, coder->component[2].sampling_factor.vertical); \
        } \
        if ( PIXEL_FORMAT == GPUJPEG_U8 ) { \
            return &gpujpeg_preprocessor_comp_to_raw_kernel<color_space_internal, COLOR, GPUJPEG_U8, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC>; \
        } else if ( PIXEL_FORMAT == GPUJPEG_444_U8_P012 ) { \
            return &gpujpeg_preprocessor_comp_to_raw_kernel<color_space_internal, COLOR, GPUJPEG_444_U8_P012, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC>; \
        } else if ( PIXEL_FORMAT == GPUJPEG_444_U8_P012Z ) { \
            return &gpujpeg_preprocessor_comp_to_raw_kernel<color_space_internal, COLOR, GPUJPEG_444_U8_P012Z, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC>; \
        } else if ( coder->param_image.pixel_format == GPUJPEG_422_U8_P1020 ) { \
            return &gpujpeg_preprocessor_comp_to_raw_kernel<color_space_internal, COLOR, GPUJPEG_422_U8_P1020, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC>; \
        } else if ( coder->param_image.pixel_format == GPUJPEG_444_U8_P0P1P2 ) { \
            return &gpujpeg_preprocessor_comp_to_raw_kernel<color_space_internal, COLOR, GPUJPEG_444_U8_P0P1P2, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC>; \
        } else if ( coder->param_image.pixel_format == GPUJPEG_422_U8_P0P1P2 ) { \
            return &gpujpeg_preprocessor_comp_to_raw_kernel<color_space_internal, COLOR, GPUJPEG_422_U8_P0P1P2, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC>; \
        } else { \
            assert(false); \
        } \
    } \

    // None color space
    if ( coder->param_image.color_space == GPUJPEG_NONE ) {
        RETURN_KERNEL(coder->param_image.pixel_format, GPUJPEG_NONE)
    }
    // RGB color space
    else if ( coder->param_image.color_space == GPUJPEG_RGB ) {
        RETURN_KERNEL(coder->param_image.pixel_format, GPUJPEG_RGB)
    }
    // YCbCr color space
    else if ( coder->param_image.color_space == GPUJPEG_YCBCR_BT601 ) {
        RETURN_KERNEL(coder->param_image.pixel_format, GPUJPEG_YCBCR_BT601)
    }
    // YCbCr color space
    else if ( coder->param_image.color_space == GPUJPEG_YCBCR_BT601_256LVLS ) {
        RETURN_KERNEL(coder->param_image.pixel_format, GPUJPEG_YCBCR_BT601_256LVLS)
    }
    // YCbCr color space
    else if ( coder->param_image.color_space == GPUJPEG_YCBCR_BT709 ) {
        RETURN_KERNEL(coder->param_image.pixel_format, GPUJPEG_YCBCR_BT709)
    }
    // YUV color space
    else if ( coder->param_image.color_space == GPUJPEG_YUV ) {
        RETURN_KERNEL(coder->param_image.pixel_format, GPUJPEG_YUV)
    }
    // Unknown color space
    else {
        assert(false);
    }

#undef RETURN_KERNEL_IF
#undef RETURN_KERNEL

    return NULL;
}

static int gpujpeg_preprocessor_decode_no_transform(struct gpujpeg_coder * coder)
{
    if (coder->param_image.comp_count == 3 && coder->param_image.color_space != coder->param.color_space_internal) {
            /*fprintf(stderr, "Decoding JPEG to a planar pixel format is supported only when no color transformation is required. "
                            "JPEG internal color space is set to \"%s\", image is \"%s\".\n",
                            gpujpeg_color_space_get_name(coder->param.color_space_internal),
                            gpujpeg_color_space_get_name(coder->param_image.color_space));*/
            return 0;
    }

    const int *sampling_factors = gpujpeg_pixel_format_get_sampling_factor(coder->param_image.pixel_format);
    for (int i = 0; i < coder->param_image.comp_count; ++i) {
        if (coder->component[i].sampling_factor.horizontal != sampling_factors[i * 2]
                || coder->component[i].sampling_factor.vertical != sampling_factors[i * 2 + 1]) {
            const char *name = gpujpeg_pixel_format_get_name(coder->param_image.pixel_format);
            /*fprintf(stderr, "Decoding JPEG to a planar pixel format cannot change subsampling (%s to %s).\n",
                    gpujpeg_subsampling_get_name(coder->param_image.comp_count, coder->component),
                    gpujpeg_pixel_format_get_name(coder->param_image.pixel_format));*/
            return 0;
        }
    }
    return 1;
}

/* Documented at declaration */
int
gpujpeg_preprocessor_decoder_init(struct gpujpeg_coder* coder)
{
    coder->preprocessor = NULL;

    if (!gpujpeg_pixel_format_is_interleaved(coder->param_image.pixel_format) &&
            gpujpeg_preprocessor_decode_no_transform(coder)) {
        if ( coder->param.verbose >= 1 ) {
            printf("Matching format detected - not using postprocessor, using memcpy instead.");
        }
        return 0;
    }

    assert(coder->param_image.comp_count == 3);

    if (coder->param.color_space_internal == GPUJPEG_NONE) {
        coder->preprocessor = (void*)gpujpeg_preprocessor_select_decode_kernel<GPUJPEG_NONE>(coder);
    }
    else if (coder->param.color_space_internal == GPUJPEG_RGB) {
        coder->preprocessor = (void*)gpujpeg_preprocessor_select_decode_kernel<GPUJPEG_RGB>(coder);
    }
    else if (coder->param.color_space_internal == GPUJPEG_YCBCR_BT601) {
        coder->preprocessor = (void*)gpujpeg_preprocessor_select_decode_kernel<GPUJPEG_YCBCR_BT601>(coder);
    }
    else if (coder->param.color_space_internal == GPUJPEG_YCBCR_BT601_256LVLS) {
        coder->preprocessor = (void*)gpujpeg_preprocessor_select_decode_kernel<GPUJPEG_YCBCR_BT601_256LVLS>(coder);
    }
    else if (coder->param.color_space_internal == GPUJPEG_YCBCR_BT709) {
        coder->preprocessor = (void*)gpujpeg_preprocessor_select_decode_kernel<GPUJPEG_YCBCR_BT709>(coder);
    }
    else {
        assert(false);
    }
    if (coder->preprocessor == NULL) {
        return -1;
    }
    return 0;
}

/**
 * Copies raw data GPU memory without running any postprocessor kernel.
 *
 * This assumes that the JPEG has same color space as input raw image and
 * currently also that the component subsampling correspond between raw and
 * JPEG (although at least different horizontal subsampling can be quite
 * easily done).
 *
 * @invariant gpujpeg_preprocessor_decode_no_transform(coder) != 0
 */
static int
gpujpeg_preprocessor_decoder_copy_planar_data(struct gpujpeg_coder * coder, hipStream_t stream)
{
    assert(coder->param_image.comp_count == 1 ||
            coder->param_image.comp_count == 3);
    size_t data_raw_offset = 0;
    bool needs_stride = false; // true if width is not divisible by MCU width
    for (int i = 0; i < coder->param_image.comp_count; ++i) {
        needs_stride = needs_stride || coder->component[i].width != coder->component[i].data_width;
    }
    if (!needs_stride) {
            for (int i = 0; i < coder->param_image.comp_count; ++i) {
                    size_t component_size = coder->component[i].width * coder->component[i].height;
                    hipMemcpyAsync(coder->d_data_raw + data_raw_offset, coder->component[i].d_data, component_size, hipMemcpyDeviceToDevice, stream);
                    data_raw_offset += component_size;
            }
    } else {
            for (int i = 0; i < coder->param_image.comp_count; ++i) {
                    int spitch = coder->component[i].data_width;
                    int dpitch = coder->component[i].width;
                    size_t component_size = spitch * coder->component[i].height;
                    hipMemcpy2DAsync(coder->d_data_raw + data_raw_offset, dpitch, coder->component[i].d_data, spitch, coder->component[i].width, coder->component[i].height, hipMemcpyDeviceToDevice, stream);
                    data_raw_offset += component_size;
            }
    }
    gpujpeg_cuda_check_error("Preprocessor copy failed", return -1);
    return 0;
}

/* Documented at declaration */
int
gpujpeg_preprocessor_decode(struct gpujpeg_coder* coder, hipStream_t stream)
{
    if (!coder->preprocessor) {
        return gpujpeg_preprocessor_decoder_copy_planar_data(coder, stream);
    }

    assert(coder->param_image.comp_count == 3);

    hipMemsetAsync(coder->d_data_raw, 0, coder->data_raw_size * sizeof(uint8_t), stream);

    // Select kernel
    gpujpeg_preprocessor_decode_kernel kernel = (gpujpeg_preprocessor_decode_kernel)coder->preprocessor;
    assert(kernel != NULL);

    int image_width = coder->param_image.width;
    int image_height = coder->param_image.height;

    // When saving 4:2:2 data of odd width, the data should have even width, so round it
    if (coder->param_image.pixel_format == GPUJPEG_422_U8_P1020) {
        image_width = gpujpeg_div_and_round_up(coder->param_image.width, 2) * 2;
    }

    // Prepare unit size
    /// @todo this stuff doesn't look correct - we multiply by unitSize and then divide by it
    int unitSize = gpujpeg_pixel_format_get_unit_size(coder->param_image.pixel_format);
    if (unitSize == 0) {
        unitSize = 1;
    }

    // Prepare kernel
    int alignedSize = gpujpeg_div_and_round_up(image_width * image_height, RGB_8BIT_THREADS) * RGB_8BIT_THREADS * unitSize;
    dim3 threads (RGB_8BIT_THREADS);
    dim3 grid (alignedSize / (RGB_8BIT_THREADS * unitSize));
    assert(alignedSize % (RGB_8BIT_THREADS * unitSize) == 0);
    if ( grid.x > GPUJPEG_CUDA_MAXIMUM_GRID_SIZE ) {
        grid.y = gpujpeg_div_and_round_up(grid.x, GPUJPEG_CUDA_MAXIMUM_GRID_SIZE);
        grid.x = GPUJPEG_CUDA_MAXIMUM_GRID_SIZE;
    }

    // Run kernel
    struct gpujpeg_preprocessor_data data;
    for ( int comp = 0; comp < 3; comp++ ) {
        assert(coder->sampling_factor.horizontal % coder->component[comp].sampling_factor.horizontal == 0);
        assert(coder->sampling_factor.vertical % coder->component[comp].sampling_factor.vertical == 0);
        data.comp[comp].d_data = coder->component[comp].d_data;
        data.comp[comp].sampling_factor.horizontal = coder->sampling_factor.horizontal / coder->component[comp].sampling_factor.horizontal;
        data.comp[comp].sampling_factor.vertical = coder->sampling_factor.vertical / coder->component[comp].sampling_factor.vertical;
        data.comp[comp].data_width = coder->component[comp].data_width;
    }
    kernel<<<grid, threads, 0, stream>>>(
        data,
        coder->d_data_raw,
        image_width,
        image_height
    );
    gpujpeg_cuda_check_error("Preprocessor encoding failed", return -1);

    return 0;
}

/* vi: set expandtab sw=4: */
