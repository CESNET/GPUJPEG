#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011-2020, CESNET z.s.p.o
 * Copyright (c) 2011, Silicon Genome, LLC.
 *
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */
/**
 * @file
 * @brief
 * This file contains preprocessors from raw image to a common format for
 * computational kernels. It also does color space transformations.
 */

#include "gpujpeg_colorspace.h"
#include "gpujpeg_preprocessor_common.h"
#include "gpujpeg_preprocessor.h"
#include "gpujpeg_util.h"

/**
 * Store value to component data buffer in specified position by buffer size and subsampling
 */
template<
    unsigned int s_samp_factor_h,
    unsigned int s_samp_factor_v
>
static __device__ void
gpujpeg_preprocessor_raw_to_comp_store_comp(uint8_t value, unsigned int position_x, unsigned int position_y, struct gpujpeg_preprocessor_data_component & comp)
{
    const unsigned int samp_factor_h = ( s_samp_factor_h == GPUJPEG_DYNAMIC ) ? comp.sampling_factor.horizontal : s_samp_factor_h;
    const unsigned int samp_factor_v = ( s_samp_factor_v == GPUJPEG_DYNAMIC ) ? comp.sampling_factor.vertical : s_samp_factor_v;

    if ( (position_x % samp_factor_h) || (position_y % samp_factor_v) )
        return;

    position_x = position_x / samp_factor_h;
    position_y = position_y / samp_factor_v;

    const unsigned int data_position = position_y * comp.data_width + position_x;
    comp.d_data[data_position] = value;
}

template<
    enum gpujpeg_pixel_format pixel_format,
    uint8_t s_comp1_samp_factor_h, uint8_t s_comp1_samp_factor_v,
    uint8_t s_comp2_samp_factor_h, uint8_t s_comp2_samp_factor_v,
    uint8_t s_comp3_samp_factor_h, uint8_t s_comp3_samp_factor_v
>
struct gpujpeg_preprocessor_raw_to_comp_store {
    static __device__ void perform(uchar4 value, unsigned int position_x, unsigned int position_y, struct gpujpeg_preprocessor_data & data) {
        gpujpeg_preprocessor_raw_to_comp_store_comp<s_comp1_samp_factor_h, s_comp1_samp_factor_v>(value.x, position_x, position_y, data.comp[0]);
        gpujpeg_preprocessor_raw_to_comp_store_comp<s_comp2_samp_factor_h, s_comp2_samp_factor_v>(value.y, position_x, position_y, data.comp[1]);
        gpujpeg_preprocessor_raw_to_comp_store_comp<s_comp3_samp_factor_h, s_comp3_samp_factor_v>(value.z, position_x, position_y, data.comp[2]);
    }
};

template<
    uint8_t s_comp1_samp_factor_h, uint8_t s_comp1_samp_factor_v,
    uint8_t s_comp2_samp_factor_h, uint8_t s_comp2_samp_factor_v,
    uint8_t s_comp3_samp_factor_h, uint8_t s_comp3_samp_factor_v
>
struct gpujpeg_preprocessor_raw_to_comp_store<GPUJPEG_444_U8_P012A, s_comp1_samp_factor_h, s_comp1_samp_factor_v, s_comp2_samp_factor_h, s_comp2_samp_factor_v, s_comp3_samp_factor_h, s_comp3_samp_factor_v> {
    static __device__ void perform (uchar4 value, unsigned int position_x, unsigned int position_y, struct gpujpeg_preprocessor_data & data) {
        gpujpeg_preprocessor_raw_to_comp_store_comp<s_comp1_samp_factor_h, s_comp1_samp_factor_v>(value.x, position_x, position_y, data.comp[0]);
        gpujpeg_preprocessor_raw_to_comp_store_comp<s_comp2_samp_factor_h, s_comp2_samp_factor_v>(value.y, position_x, position_y, data.comp[1]);
        gpujpeg_preprocessor_raw_to_comp_store_comp<s_comp3_samp_factor_h, s_comp3_samp_factor_v>(value.z, position_x, position_y, data.comp[2]);
        gpujpeg_preprocessor_raw_to_comp_store_comp<s_comp1_samp_factor_h, s_comp1_samp_factor_v>(value.w, position_x, position_y, data.comp[3]);
    }
};

template<enum gpujpeg_pixel_format>
inline __device__ void raw_to_comp_load(const uint8_t* d_data_raw, int &image_width, int &image_height, int &image_position, int &x, int &y, uchar4 &r);

template<>
inline __device__ void raw_to_comp_load<GPUJPEG_U8>(const uint8_t* d_data_raw, int &image_width, int &image_height, int &image_position, int &x, int &y, uchar4 &r)
{
    r.x = d_data_raw[image_position];
    r.y = 128;
    r.z = 128;
}

template<>
inline __device__ void raw_to_comp_load<GPUJPEG_444_U8_P0P1P2>(const uint8_t* d_data_raw, int &image_width, int &image_height, int &image_position, int &x, int &y, uchar4 &r)
{
    r.x = d_data_raw[image_position];
    r.y = d_data_raw[image_width * image_height + image_position];
    r.z = d_data_raw[2 * image_width * image_height + image_position];
}

template<>
inline __device__ void raw_to_comp_load<GPUJPEG_422_U8_P0P1P2>(const uint8_t* d_data_raw, int &image_width, int &image_height, int &image_position, int &x, int &y, uchar4 &r)
{
    r.x = d_data_raw[image_position];
    r.y = d_data_raw[image_width * image_height + image_position / 2];
    r.z = d_data_raw[image_width * image_height + image_height * ((image_width + 1) / 2) + image_position / 2];
}

template<>
inline __device__ void raw_to_comp_load<GPUJPEG_420_U8_P0P1P2>(const uint8_t* d_data_raw, int &image_width, int &image_height, int &image_position, int &x, int &y, uchar4 &r)
{
    r.x = d_data_raw[image_position];
    r.y = d_data_raw[image_width * image_height + y / 2 * ((image_width + 1) / 2) + x / 2];
    r.z = d_data_raw[image_width * image_height + ((image_height + 1) / 2 + y / 2) * ((image_width + 1) / 2) + x / 2];
}

template<>
inline __device__ void raw_to_comp_load<GPUJPEG_444_U8_P012>(const uint8_t* d_data_raw, int &image_width, int &image_height, int &image_position, int &x, int &y, uchar4 &r)
{
    const unsigned int offset = image_position * 3;
    r.x = d_data_raw[offset];
    r.y = d_data_raw[offset + 1];
    r.z = d_data_raw[offset + 2];
}

template<>
inline __device__ void raw_to_comp_load<GPUJPEG_444_U8_P012A>(const uint8_t* d_data_raw, int &image_width, int &image_height, int &image_position, int &x, int &y, uchar4 &r)
{
    const unsigned int offset = image_position * 4;
    r.x = d_data_raw[offset];
    r.y = d_data_raw[offset + 1];
    r.z = d_data_raw[offset + 2];
    r.w = d_data_raw[offset + 3];
}

template<>
inline __device__ void raw_to_comp_load<GPUJPEG_444_U8_P012Z>(const uint8_t* d_data_raw, int &image_width, int &image_height, int &image_position, int &x, int &y, uchar4 &r)
{
    const unsigned int offset = image_position * 4;
    r.x = d_data_raw[offset];
    r.y = d_data_raw[offset + 1];
    r.z = d_data_raw[offset + 2];
}

template<>
inline __device__ void raw_to_comp_load<GPUJPEG_422_U8_P1020>(const uint8_t* d_data_raw, int &image_width, int &image_height, int &image_position, int &x, int &y, uchar4 &r)
{
    const unsigned int offset = image_position * 2;
    r.x = d_data_raw[offset + 1];
    if ( image_position % 2 == 0 ) {
        r.y = d_data_raw[offset];
        r.z = d_data_raw[offset + 2];
    } else {
        r.y = d_data_raw[offset - 2];
        r.z = d_data_raw[offset];
    }
}

/**
 * Kernel - Copy raw image source data into three separated component buffers
 */
typedef void (*gpujpeg_preprocessor_encode_kernel)(struct gpujpeg_preprocessor_data data, const uint8_t* d_data_raw, const uint8_t* d_data_raw_end, int image_width, int image_height, uint32_t width_div_mul, uint32_t width_div_shift);

/**
 * @note
 * In previous versions, there was an optimalization with aligned preloading to shared memory.
 * This was, however, removed because it didn't exhibit any performance improvement anymore
 * (actually removing that yields slight performance gain).
 */
template<
    enum gpujpeg_color_space color_space_internal,
    enum gpujpeg_color_space color_space,
    enum gpujpeg_pixel_format pixel_format,
    uint8_t s_comp1_samp_factor_h, uint8_t s_comp1_samp_factor_v,
    uint8_t s_comp2_samp_factor_h, uint8_t s_comp2_samp_factor_v,
    uint8_t s_comp3_samp_factor_h, uint8_t s_comp3_samp_factor_v
>
__global__ void
gpujpeg_preprocessor_raw_to_comp_kernel(struct gpujpeg_preprocessor_data data, const uint8_t* d_data_raw, const uint8_t* d_data_raw_end, int image_width, int image_height, uint32_t width_div_mul, uint32_t width_div_shift)
{
    int x  = threadIdx.x;
    int gX = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;

    // Position
    int image_position = gX + x;
    int image_position_y = gpujpeg_const_div_divide(image_position, width_div_mul, width_div_shift);
    int image_position_x = image_position - (image_position_y * image_width);

    // Load
    uchar4 r;
    raw_to_comp_load<pixel_format>(d_data_raw, image_width, image_height, image_position, image_position_x, image_position_y, r);

    // Color transform
    gpujpeg_color_transform<color_space, color_space_internal>::perform(r);

    // Store
    if ( image_position < (image_width * image_height) ) {
        gpujpeg_preprocessor_raw_to_comp_store<pixel_format, s_comp1_samp_factor_h, s_comp1_samp_factor_v, s_comp2_samp_factor_h, s_comp2_samp_factor_v, s_comp3_samp_factor_h, s_comp3_samp_factor_v>::perform(r, image_position_x, image_position_y, data);
    }
}

/**
 * Select preprocessor encode kernel
 *
 * @param encoder
 * @return kernel
 */
template<enum gpujpeg_color_space color_space_internal>
gpujpeg_preprocessor_encode_kernel
gpujpeg_preprocessor_select_encode_kernel(struct gpujpeg_coder* coder)
{
    gpujpeg_preprocessor_sampling_factor_t sampling_factor = gpujpeg_preprocessor_make_sampling_factor(
        coder->sampling_factor.horizontal / coder->component[0].sampling_factor.horizontal,
        coder->sampling_factor.vertical / coder->component[0].sampling_factor.vertical,
        coder->sampling_factor.horizontal / coder->component[1].sampling_factor.horizontal,
        coder->sampling_factor.vertical / coder->component[1].sampling_factor.vertical,
        coder->sampling_factor.horizontal / coder->component[2].sampling_factor.horizontal,
        coder->sampling_factor.vertical / coder->component[2].sampling_factor.vertical
    );

    /// @todo allow also different susbsampling for 4rd channel than for first
    assert(coder->param_image.comp_count != 4 ||
            (coder->component[0].sampling_factor.horizontal == coder->component[3].sampling_factor.horizontal &&
             coder->component[0].sampling_factor.vertical == coder->component[3].sampling_factor.vertical));

#define RETURN_KERNEL_IF(PIXEL_FORMAT, COLOR, P1, P2, P3, P4, P5, P6) \
    if ( sampling_factor == gpujpeg_preprocessor_make_sampling_factor(P1, P2, P3, P4, P5, P6) ) { \
        int max_h = max(P1, max(P3, P5)); \
        int max_v = max(P2, max(P4, P6)); \
        if ( coder->param.verbose >= 1 ) { \
            printf("Using faster kernel for preprocessor (precompiled %dx%d, %dx%d, %dx%d).\n", max_h / P1, max_v / P2, max_h / P3, max_v / P4, max_h / P5, max_v / P6); \
        } \
        switch ( PIXEL_FORMAT ) { \
            case GPUJPEG_444_U8_P012: return &gpujpeg_preprocessor_raw_to_comp_kernel<color_space_internal, COLOR, GPUJPEG_444_U8_P012, P1, P2, P3, P4, P5, P6>; \
            case GPUJPEG_444_U8_P012A: return coder->param_image.comp_count == 4 ? &gpujpeg_preprocessor_raw_to_comp_kernel<color_space_internal, COLOR, GPUJPEG_444_U8_P012A, P1, P2, P3, P4, P5, P6> : &gpujpeg_preprocessor_raw_to_comp_kernel<color_space_internal, COLOR, GPUJPEG_444_U8_P012Z, P1, P2, P3, P4, P5, P6>; \
            case GPUJPEG_444_U8_P012Z: return  &gpujpeg_preprocessor_raw_to_comp_kernel<color_space_internal, COLOR, GPUJPEG_444_U8_P012Z, P1, P2, P3, P4, P5, P6>; \
            case GPUJPEG_422_U8_P1020: return &gpujpeg_preprocessor_raw_to_comp_kernel<color_space_internal, COLOR, GPUJPEG_422_U8_P1020, P1, P2, P3, P4, P5, P6>; \
            case GPUJPEG_444_U8_P0P1P2: return &gpujpeg_preprocessor_raw_to_comp_kernel<color_space_internal, COLOR, GPUJPEG_444_U8_P0P1P2, P1, P2, P3, P4, P5, P6>; \
            case GPUJPEG_422_U8_P0P1P2: return &gpujpeg_preprocessor_raw_to_comp_kernel<color_space_internal, COLOR, GPUJPEG_422_U8_P0P1P2, P1, P2, P3, P4, P5, P6>; \
            case GPUJPEG_420_U8_P0P1P2: return &gpujpeg_preprocessor_raw_to_comp_kernel<color_space_internal, COLOR, GPUJPEG_420_U8_P0P1P2, P1, P2, P3, P4, P5, P6>; \
            case GPUJPEG_U8: return &gpujpeg_preprocessor_raw_to_comp_kernel<color_space_internal, COLOR, GPUJPEG_U8, P1, P2, P3, P4, P5, P6>; \
            case GPUJPEG_PIXFMT_NONE: abort(); \
        } \
    }
#define RETURN_KERNEL(PIXEL_FORMAT, COLOR) \
    RETURN_KERNEL_IF(PIXEL_FORMAT, COLOR, 1, 1, 1, 1, 1, 1) /* 4:4:4 */ \
    else RETURN_KERNEL_IF(PIXEL_FORMAT, COLOR, 1, 1, 2, 2, 2, 2) /* 4:2:0 */ \
    else RETURN_KERNEL_IF(PIXEL_FORMAT, COLOR, 1, 1, 1, 2, 1, 2) /* 4:4:0 */ \
    else RETURN_KERNEL_IF(PIXEL_FORMAT, COLOR, 1, 1, 2, 1, 2, 1) /* 4:2:2 */ \
    else { \
        if ( coder->param.verbose >= 0 ) { \
            printf("Using slower kernel for preprocessor (dynamic %dx%d, %dx%d, %dx%d).\n", coder->component[0].sampling_factor.horizontal, coder->component[0].sampling_factor.vertical, coder->component[1].sampling_factor.horizontal, coder->component[1].sampling_factor.vertical, coder->component[2].sampling_factor.horizontal, coder->component[2].sampling_factor.vertical); \
        } \
        switch ( PIXEL_FORMAT ) { \
            case GPUJPEG_444_U8_P012: return &gpujpeg_preprocessor_raw_to_comp_kernel<color_space_internal, COLOR, GPUJPEG_444_U8_P012, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC>; \
            case GPUJPEG_444_U8_P012A: return coder->param_image.comp_count == 4 ? &gpujpeg_preprocessor_raw_to_comp_kernel<color_space_internal, COLOR, GPUJPEG_444_U8_P012A, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC> : &gpujpeg_preprocessor_raw_to_comp_kernel<color_space_internal, COLOR, GPUJPEG_444_U8_P012Z, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC>; \
            case GPUJPEG_444_U8_P012Z: return &gpujpeg_preprocessor_raw_to_comp_kernel<color_space_internal, COLOR, GPUJPEG_444_U8_P012Z, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC>; \
            case GPUJPEG_422_U8_P1020: return &gpujpeg_preprocessor_raw_to_comp_kernel<color_space_internal, COLOR, GPUJPEG_422_U8_P1020, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC>; \
            case GPUJPEG_444_U8_P0P1P2: return &gpujpeg_preprocessor_raw_to_comp_kernel<color_space_internal, COLOR, GPUJPEG_444_U8_P0P1P2, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC>; \
            case GPUJPEG_422_U8_P0P1P2: return &gpujpeg_preprocessor_raw_to_comp_kernel<color_space_internal, COLOR, GPUJPEG_422_U8_P0P1P2, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC>; \
            case GPUJPEG_420_U8_P0P1P2: return &gpujpeg_preprocessor_raw_to_comp_kernel<color_space_internal, COLOR, GPUJPEG_420_U8_P0P1P2, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC>; \
            case GPUJPEG_U8: return &gpujpeg_preprocessor_raw_to_comp_kernel<color_space_internal, COLOR, GPUJPEG_U8, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC>; \
            case GPUJPEG_PIXFMT_NONE: abort(); \
        } \
    } \

    // None color space
    if ( coder->param_image.color_space == GPUJPEG_NONE ) {
        RETURN_KERNEL(coder->param_image.pixel_format, GPUJPEG_NONE);
    }
    // RGB color space
    else if ( coder->param_image.color_space == GPUJPEG_RGB ) {
        RETURN_KERNEL(coder->param_image.pixel_format, GPUJPEG_RGB);
    }
    // YCbCr color space
    else if ( coder->param_image.color_space == GPUJPEG_YCBCR_BT601 ) {
        RETURN_KERNEL(coder->param_image.pixel_format, GPUJPEG_YCBCR_BT601);
    }
    // YCbCr color space
    else if ( coder->param_image.color_space == GPUJPEG_YCBCR_BT601_256LVLS ) {
        RETURN_KERNEL(coder->param_image.pixel_format, GPUJPEG_YCBCR_BT601_256LVLS);
    }
    // YCbCr color space
    else if ( coder->param_image.color_space == GPUJPEG_YCBCR_BT709 ) {
        RETURN_KERNEL(coder->param_image.pixel_format, GPUJPEG_YCBCR_BT709);
    }
#ifdef ENABLE_YUV
    // YUV color space
    else if ( coder->param_image.color_space == GPUJPEG_YUV ) {
        RETURN_KERNEL(coder->param_image.pixel_format, GPUJPEG_YUV);
    }
#endif
    // Unknown color space
    else {
        assert(false);
    }

#undef RETURN_KERNEL_IF
#undef RETURN_KERNEL

    return NULL;
}

static int gpujpeg_preprocessor_encode_no_transform(struct gpujpeg_coder * coder)
{
    if (gpujpeg_pixel_format_is_interleaved(coder->param_image.pixel_format)) {
        return 0;
    }

    if (coder->param_image.comp_count == 3 && coder->param_image.color_space != coder->param.color_space_internal) {
        return 0;
    }

    const int *sampling_factors = gpujpeg_pixel_format_get_sampling_factor(coder->param_image.pixel_format);
    for (int i = 0; i < coder->param_image.comp_count; ++i) {
        if (coder->component[i].sampling_factor.horizontal != sampling_factors[i * 2]
                || coder->component[i].sampling_factor.vertical != sampling_factors[i * 2 + 1]) {
            return 0;
        }
    }
    return 1;
}

/* Documented at declaration */
int
gpujpeg_preprocessor_encoder_init(struct gpujpeg_coder* coder)
{
    coder->preprocessor = NULL;

    if ( coder->param_image.comp_count == 1 ) {
        return 0;
    }

    if ( gpujpeg_preprocessor_encode_no_transform(coder) ) {
        if ( coder->param.verbose >= 2 ) {
            printf("Matching format detected - not using preprocessor, using memcpy instead.");
        }
        return 0;
    }

    assert(coder->param_image.comp_count == 3 || coder->param_image.comp_count == 4);

    if (coder->param.color_space_internal == GPUJPEG_NONE) {
        coder->preprocessor = (void*)gpujpeg_preprocessor_select_encode_kernel<GPUJPEG_NONE>(coder);
    }
    else if (coder->param.color_space_internal == GPUJPEG_RGB) {
        coder->preprocessor = (void*)gpujpeg_preprocessor_select_encode_kernel<GPUJPEG_RGB>(coder);
    }
    else if (coder->param.color_space_internal == GPUJPEG_YCBCR_BT601) {
        coder->preprocessor = (void*)gpujpeg_preprocessor_select_encode_kernel<GPUJPEG_YCBCR_BT601>(coder);
    }
    else if (coder->param.color_space_internal == GPUJPEG_YCBCR_BT601_256LVLS) {
        coder->preprocessor = (void*)gpujpeg_preprocessor_select_encode_kernel<GPUJPEG_YCBCR_BT601_256LVLS>(coder);
    }
    else if (coder->param.color_space_internal == GPUJPEG_YCBCR_BT709) {
        coder->preprocessor = (void*)gpujpeg_preprocessor_select_encode_kernel<GPUJPEG_YCBCR_BT709>(coder);
    }

    if ( coder->preprocessor == NULL ) {
        return -1;
    }

    return 0;
}

int
gpujpeg_preprocessor_encode_interlaced(struct gpujpeg_encoder * encoder)
{
    struct gpujpeg_coder* coder = &encoder->coder;

    hipMemsetAsync(coder->d_data, 0, coder->data_size * sizeof(uint8_t), encoder->stream);
    gpujpeg_cuda_check_error("Preprocessor memset failed", return -1);

    // Select kernel
    gpujpeg_preprocessor_encode_kernel kernel = (gpujpeg_preprocessor_encode_kernel) coder->preprocessor;
    assert(kernel != NULL);

    int image_width = coder->param_image.width;
    int image_height = coder->param_image.height;

    // When loading 4:2:2 data of odd width, the data in fact has even width, so round it
    // (at least imagemagick convert tool generates data stream in this way)
    if (coder->param_image.pixel_format == GPUJPEG_422_U8_P1020) {
        image_width = (coder->param_image.width + 1) & ~1;
    }

    // Prepare unit size
    /// @todo this stuff doesn't look correct - we multiply by unitSize and then divide by it
    int unitSize = gpujpeg_pixel_format_get_unit_size(coder->param_image.pixel_format);
    if (unitSize == 0) {
        unitSize = 1;
    }

    // Prepare kernel
    int alignedSize = gpujpeg_div_and_round_up(image_width * image_height, RGB_8BIT_THREADS) * RGB_8BIT_THREADS * unitSize;
    dim3 threads (RGB_8BIT_THREADS);
    dim3 grid (alignedSize / (RGB_8BIT_THREADS * unitSize));
    assert(alignedSize % (RGB_8BIT_THREADS * unitSize) == 0);
    while ( grid.x > GPUJPEG_CUDA_MAXIMUM_GRID_SIZE ) {
        grid.y *= 2;
        grid.x = gpujpeg_div_and_round_up(grid.x, 2);
    }

    // Decompose input image width for faster division using multiply-high and right shift
    uint32_t width_div_mul, width_div_shift;
    gpujpeg_const_div_prepare(image_width, width_div_mul, width_div_shift);

    // Run kernel
    struct gpujpeg_preprocessor_data data;
    for ( int comp = 0; comp < coder->param_image.comp_count; comp++ ) {
        assert(coder->sampling_factor.horizontal % coder->component[comp].sampling_factor.horizontal == 0);
        assert(coder->sampling_factor.vertical % coder->component[comp].sampling_factor.vertical == 0);
        data.comp[comp].d_data = coder->component[comp].d_data;
        data.comp[comp].sampling_factor.horizontal = coder->sampling_factor.horizontal / coder->component[comp].sampling_factor.horizontal;
        data.comp[comp].sampling_factor.vertical = coder->sampling_factor.vertical / coder->component[comp].sampling_factor.vertical;
        data.comp[comp].data_width = coder->component[comp].data_width;
    }
    kernel<<<grid, threads, 0, encoder->stream>>>(
        data,
        coder->d_data_raw,
        coder->d_data_raw + coder->data_raw_size,
        image_width,
        image_height,
        width_div_mul,
        width_div_shift
    );
    gpujpeg_cuda_check_error("Preprocessor encoding failed", return -1);

    return 0;
}

/**
 * Copies raw data from source image to GPU memory without running
 * any preprocessor kernel.
 *
 * This assumes that the JPEG has same color space as input raw image and
 * currently also that the component subsampling correspond between raw and
 * JPEG (although at least different horizontal subsampling can be quite
 * easily done).
 *
 * @invariant gpujpeg_preprocessor_encode_no_transform(coder) != 0
 */
static int
gpujpeg_preprocessor_encoder_copy_planar_data(struct gpujpeg_encoder * encoder)
{
    struct gpujpeg_coder * coder = &encoder->coder;
    assert(coder->param_image.comp_count == 1 ||
            coder->param_image.comp_count == 3);

    size_t data_raw_offset = 0;
    bool needs_stride = false; // true if width is not divisible by MCU width
    for (int i = 0; i < coder->param_image.comp_count; ++i) {
        needs_stride = needs_stride || coder->component[i].width != coder->component[i].data_width;
    }
    if (!needs_stride) {
            for (int i = 0; i < coder->param_image.comp_count; ++i) {
                    size_t component_size = coder->component[i].width * coder->component[i].height;
                    hipMemcpyAsync(coder->component[i].d_data, coder->d_data_raw + data_raw_offset, component_size, hipMemcpyDeviceToDevice, encoder->stream);
                    data_raw_offset += component_size;
            }
    } else {
            for (int i = 0; i < coder->param_image.comp_count; ++i) {
                    int spitch = coder->component[i].width;
                    int dpitch = coder->component[i].data_width;
                    size_t component_size = spitch * coder->component[i].height;
                    hipMemcpy2DAsync(coder->component[i].d_data, dpitch, coder->d_data_raw + data_raw_offset, spitch, spitch, coder->component[i].height, hipMemcpyDeviceToDevice, encoder->stream);
                    data_raw_offset += component_size;
            }
    }
    gpujpeg_cuda_check_error("Preprocessor copy failed", return -1);
    return 0;
}

/* Documented at declaration */
int
gpujpeg_preprocessor_encode(struct gpujpeg_encoder * encoder)
{
    struct gpujpeg_coder * coder = &encoder->coder;
    if (coder->preprocessor) {
            return gpujpeg_preprocessor_encode_interlaced(encoder);
    } else {
        return gpujpeg_preprocessor_encoder_copy_planar_data(encoder);
    }
}

/* vi: set expandtab sw=4: */
