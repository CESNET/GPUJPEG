#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2011, CESNET z.s.p.o
 * Copyright (c) 2011, Silicon Genome, LLC.
 *
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */
 
#include "gpujpeg_preprocessor.h"
#include <libgpujpeg/gpujpeg_util.h>
#include "gpujpeg_colorspace.h"

#define RGB_8BIT_THREADS 256

/**
 * Preprocessor data for component
 */
struct gpujpeg_preprocessor_data_component
{
    uint8_t* d_data;
    int data_width;
    struct gpujpeg_component_sampling_factor sampling_factor;
};

/**
 * Preprocessor data
 */
struct gpujpeg_preprocessor_data
{
    struct gpujpeg_preprocessor_data_component comp[3];
};

/** Value that means that sampling factor has dynamic value */
#define GPUJPEG_DYNAMIC 16

/** Sampling factor for all components */
typedef int gpujpeg_preprocessor_sampling_factor_t;

/**
 * Prepares fixed divisor for dividing unsigned integers up to 2^31 
 * with unsigned integers up to 2^31.
 * Source: http://www.hackersdelight.org/HDcode/magic.c.txt
 * Modified for positive numbers only.
 */
static void
gpujpeg_const_div_prepare(const uint32_t d, uint32_t & pre_div_mul, uint32_t & pre_div_shift) {
    if(d > 1) {
        uint32_t delta;
        const uint32_t two31 = 0x80000000; // 2**31.
        const uint32_t anc = two31 - 1 - two31 % d; // Absolute value of nc.
        int p = 31;                        // Init. p.
        uint32_t q1 = two31 / anc;         // Init. q1 = 2**p/|nc|.
        uint32_t r1 = two31 - q1 * anc;    // Init. r1 = rem(2**p, |nc|).
        uint32_t q2 = two31 / d;           // Init. q2 = 2**p/|d|.
        uint32_t r2 = two31 - q2 * d;      // Init. r2 = rem(2**p, |d|).
        do {
            p = p + 1;
            q1 = 2 * q1;                   // Update q1 = 2**p/|nc|.
            r1 = 2 * r1;                   // Update r1 = rem(2**p, |nc|).
            if (r1 >= anc) {               // (Must be an unsigned
                q1 = q1 + 1;               // comparison here).
                r1 = r1 - anc;
            }
            q2 = 2 * q2;                   // Update q2 = 2**p/|d|.
            r2 = 2 * r2;                   // Update r2 = rem(2**p, |d|).
            if (r2 >= d) {                 // (Must be an unsigned
                q2 = q2 + 1;               // comparison here).
                r2 = r2 - d;
            }
            delta = d - r2;
        } while (q1 < delta || (q1 == delta && r1 == 0));
        pre_div_mul = q2 + 1;
        pre_div_shift = p - 32;            // shift amount to return.
    } else {
        pre_div_mul = 0;                   // special case for d = 1
        pre_div_shift = 0;
    }
}


/**
 * Divides unsigned numerator (up to 2^31) by precomputed constant denominator.
 */
__device__ static uint32_t
gpujpeg_const_div_divide(const uint32_t numerator, const uint32_t pre_div_mul, const uint32_t pre_div_shift) {
    return pre_div_mul ? __umulhi(numerator, pre_div_mul) >> pre_div_shift : numerator;
}

/**
 * Compose sampling factor for all components to single type
 * 
 * @return integer that contains all sampling factors
 */
inline gpujpeg_preprocessor_sampling_factor_t
gpujpeg_preprocessor_make_sampling_factor(int comp1_h, int comp1_v, int comp2_h, int comp2_v, int comp3_h, int comp3_v)
{    
    gpujpeg_preprocessor_sampling_factor_t sampling_factor = 0;
    sampling_factor |= ((comp1_h << 4) | comp1_v) << 16;
    sampling_factor |= ((comp2_h << 4) | comp2_v) << 8;
    sampling_factor |= ((comp3_h << 4) | comp3_v) << 0;
    
    return sampling_factor;
}

/**
 * Store value to component data buffer in specified position by buffer size and subsampling
 */
template<
    unsigned int s_samp_factor_h,
    unsigned int s_samp_factor_v
>
static __device__ void
gpujpeg_preprocessor_raw_to_comp_store(uint8_t value, unsigned int position_x, unsigned int position_y, struct gpujpeg_preprocessor_data_component & comp)
{
    const unsigned int samp_factor_h = ( s_samp_factor_h == GPUJPEG_DYNAMIC ) ? comp.sampling_factor.horizontal : s_samp_factor_h;
    const unsigned int samp_factor_v = ( s_samp_factor_v == GPUJPEG_DYNAMIC ) ? comp.sampling_factor.vertical : s_samp_factor_v;
    
    if ( (position_x % samp_factor_h) || (position_y % samp_factor_v) )
        return;

    position_x = position_x / samp_factor_h;
    position_y = position_y / samp_factor_v;

    const unsigned int data_position = position_y * comp.data_width + position_x;
    comp.d_data[data_position] = value;
}

/**
 * Kernel - Copy raw image source data into three separated component buffers
 */
typedef void (*gpujpeg_preprocessor_encode_kernel)(struct gpujpeg_preprocessor_data data, const uint8_t* d_data_raw, const uint8_t* d_data_raw_end, int image_width, int image_height, uint32_t width_div_mul, uint32_t width_div_shift);
 
/** Specialization [sampling factor is 4:4:4] */
template<
    enum gpujpeg_color_space color_space_internal,
    enum gpujpeg_color_space color_space,
    uint8_t s_comp1_samp_factor_h, uint8_t s_comp1_samp_factor_v,
    uint8_t s_comp2_samp_factor_h, uint8_t s_comp2_samp_factor_v,
    uint8_t s_comp3_samp_factor_h, uint8_t s_comp3_samp_factor_v
>
__global__ void 
gpujpeg_preprocessor_raw_to_comp_kernel_4_4_4(struct gpujpeg_preprocessor_data data, const uint8_t* d_data_raw, const uint8_t* d_data_raw_end, int image_width, int image_height, uint32_t width_div_mul, uint32_t width_div_shift)
{
    int x  = threadIdx.x;
    int gX = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
            
    // Load to shared
    __shared__ unsigned char s_data[RGB_8BIT_THREADS * 3];
    if ( (x * 4) < RGB_8BIT_THREADS * 3 ) {
        uint32_t* s = (uint32_t *) d_data_raw + ((gX * 3) >> 2) + x;
        uint32_t* d = (uint32_t *) s_data + x;
        if ((uint8_t *) s < d_data_raw_end) {
            *d = *s;
        }
    }
    __syncthreads();

    // Load
    int offset = x * 3;
    uint8_t r1 = s_data[offset];
    uint8_t r2 = s_data[offset + 1];
    uint8_t r3 = s_data[offset + 2];

    // Load Order
    gpujpeg_color_order<color_space>::perform_load(r1, r2, r3);

    // Color transform
    gpujpeg_color_transform<color_space, color_space_internal>::perform(r1, r2, r3);
    
    // Position
    int image_position = gX + x;
    int image_position_y = gpujpeg_const_div_divide(image_position, width_div_mul, width_div_shift);
    int image_position_x = image_position - (image_position_y * image_width);
        
    // Store
    if ( image_position < (image_width * image_height) ) {
    
        gpujpeg_preprocessor_raw_to_comp_store<s_comp1_samp_factor_h, s_comp1_samp_factor_v>(r1, image_position_x, image_position_y, data.comp[0]);
        gpujpeg_preprocessor_raw_to_comp_store<s_comp2_samp_factor_h, s_comp2_samp_factor_v>(r2, image_position_x, image_position_y, data.comp[1]);
        gpujpeg_preprocessor_raw_to_comp_store<s_comp3_samp_factor_h, s_comp3_samp_factor_v>(r3, image_position_x, image_position_y, data.comp[2]);
    }
}

/** Specialization [sampling factor is 4:2:2] */
template<
    enum gpujpeg_color_space color_space_internal,
    enum gpujpeg_color_space color_space,
    uint8_t s_comp1_samp_factor_h, uint8_t s_comp1_samp_factor_v,
    uint8_t s_comp2_samp_factor_h, uint8_t s_comp2_samp_factor_v,
    uint8_t s_comp3_samp_factor_h, uint8_t s_comp3_samp_factor_v
>
__global__ void 
gpujpeg_preprocessor_raw_to_comp_kernel_4_2_2(struct gpujpeg_preprocessor_data data, const uint8_t* d_data_raw, const uint8_t* d_data_raw_end, int image_width, int image_height, uint32_t width_div_mul, uint32_t width_div_shift)
{
    int x  = threadIdx.x;
    int gX = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
    
    // Load to shared
    __shared__ unsigned char s_data[RGB_8BIT_THREADS * 2];
    if ( (x * 4) < RGB_8BIT_THREADS * 2 ) {
        uint32_t* s = (uint32_t *) d_data_raw + ((gX * 2) >> 2) + x;
        uint32_t* d = (uint32_t *) s_data + x;
        if ((uint8_t *) s < d_data_raw_end) {
            *d = *s;
        }
    }
    __syncthreads();

    // Load
    const unsigned int offset = x * 2;
    uint8_t r1;
    uint8_t r2 = s_data[offset + 1];
    uint8_t r3;
    if ( (gX + x) % 2 == 0 ) {
        r1 = s_data[offset];
        r3 = s_data[offset + 2];
    } else {
        r1 = s_data[offset - 2];
        r3 = s_data[offset];
    }

    // Load Order
    gpujpeg_color_order<color_space>::perform_load(r1, r2, r3);
    
    // Color transform
    gpujpeg_color_transform<color_space, color_space_internal>::perform(r1, r2, r3);
    
    // Position
    int image_position = gX + x;
    int image_position_y = gpujpeg_const_div_divide(image_position, width_div_mul, width_div_shift);
    int image_position_x = image_position - (image_position_y * image_width);
    
    // Store
    if ( image_position < (image_width * image_height) ) {
        gpujpeg_preprocessor_raw_to_comp_store<s_comp1_samp_factor_h, s_comp1_samp_factor_v>(r1, image_position_x, image_position_y, data.comp[0]);
        gpujpeg_preprocessor_raw_to_comp_store<s_comp2_samp_factor_h, s_comp2_samp_factor_v>(r2, image_position_x, image_position_y, data.comp[1]);
        gpujpeg_preprocessor_raw_to_comp_store<s_comp3_samp_factor_h, s_comp3_samp_factor_v>(r3, image_position_x, image_position_y, data.comp[2]);
    }
}

/**
 * Select preprocessor encode kernel
 * 
 * @param encoder
 * @return kernel
 */
template<enum gpujpeg_color_space color_space_internal>
gpujpeg_preprocessor_encode_kernel
gpujpeg_preprocessor_select_encode_kernel(struct gpujpeg_coder* coder)
{
    gpujpeg_preprocessor_sampling_factor_t sampling_factor = gpujpeg_preprocessor_make_sampling_factor(
        coder->sampling_factor.horizontal / coder->component[0].sampling_factor.horizontal,
        coder->sampling_factor.vertical / coder->component[0].sampling_factor.vertical,
        coder->sampling_factor.horizontal / coder->component[1].sampling_factor.horizontal,
        coder->sampling_factor.vertical / coder->component[1].sampling_factor.vertical,
        coder->sampling_factor.horizontal / coder->component[2].sampling_factor.horizontal,
        coder->sampling_factor.vertical / coder->component[2].sampling_factor.vertical
    );
    
#define RETURN_KERNEL_IF(KERNEL, COLOR, P1, P2, P3, P4, P5, P6) \
    if ( sampling_factor == gpujpeg_preprocessor_make_sampling_factor(P1, P2, P3, P4, P5, P6) ) { \
        int max_h = max(P1, max(P3, P5)); \
        int max_v = max(P2, max(P4, P6)); \
        if ( coder->param.verbose ) { \
            printf("Using faster kernel for preprocessor (precompiled %dx%d, %dx%d, %dx%d).\n", max_h / P1, max_v / P2, max_h / P3, max_v / P4, max_h / P5, max_v / P6); \
        } \
        return &KERNEL<color_space_internal, COLOR, P1, P2, P3, P4, P5, P6>; \
    } 
#define RETURN_KERNEL(KERNEL, COLOR) \
    RETURN_KERNEL_IF(KERNEL, COLOR, 1, 1, 1, 1, 1, 1) \
    else RETURN_KERNEL_IF(KERNEL, COLOR, 1, 1, 2, 2, 2, 2) \
    else RETURN_KERNEL_IF(KERNEL, COLOR, 1, 1, 1, 2, 1, 2) \
    else RETURN_KERNEL_IF(KERNEL, COLOR, 1, 1, 2, 1, 2, 1) \
    else RETURN_KERNEL_IF(KERNEL, COLOR, 1, 1, 4, 4, 4, 4) \
    else { \
        if ( coder->param.verbose ) { \
            printf("Using slower kernel for preprocessor (dynamic %dx%d, %dx%d, %dx%d).\n", coder->component[0].sampling_factor.horizontal, coder->component[0].sampling_factor.vertical, coder->component[1].sampling_factor.horizontal, coder->component[1].sampling_factor.vertical, coder->component[2].sampling_factor.horizontal, coder->component[2].sampling_factor.vertical); \
        } \
        return &KERNEL<color_space_internal, COLOR, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC>; \
    } \

    // None color space
    if ( coder->param_image.color_space == GPUJPEG_NONE ) {
        if ( coder->param_image.sampling_factor == GPUJPEG_4_4_4 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_raw_to_comp_kernel_4_4_4, GPUJPEG_NONE);
        } else if ( coder->param_image.sampling_factor == GPUJPEG_4_2_2 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_raw_to_comp_kernel_4_2_2, GPUJPEG_NONE);
        } else {
            assert(false);
        }
    }// RGB color space
    else if ( coder->param_image.color_space == GPUJPEG_RGB ) {
        if ( coder->param_image.sampling_factor == GPUJPEG_4_4_4 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_raw_to_comp_kernel_4_4_4, GPUJPEG_RGB);
        } else if ( coder->param_image.sampling_factor == GPUJPEG_4_2_2 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_raw_to_comp_kernel_4_2_2, GPUJPEG_RGB);
        } else {
            assert(false);
        }
    } 
    // YCbCr color space
    else if ( coder->param_image.color_space == GPUJPEG_YCBCR_BT601 ) {
        if ( coder->param_image.sampling_factor == GPUJPEG_4_4_4 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_raw_to_comp_kernel_4_4_4, GPUJPEG_YCBCR_BT601);
        } else if ( coder->param_image.sampling_factor == GPUJPEG_4_2_2 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_raw_to_comp_kernel_4_2_2, GPUJPEG_YCBCR_BT601);
        } else {
            assert(false);
        }
    } 
    // YCbCr color space
    else if ( coder->param_image.color_space == GPUJPEG_YCBCR_BT601_256LVLS ) {
        if ( coder->param_image.sampling_factor == GPUJPEG_4_4_4 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_raw_to_comp_kernel_4_4_4, GPUJPEG_YCBCR_BT601_256LVLS);
        } else if ( coder->param_image.sampling_factor == GPUJPEG_4_2_2 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_raw_to_comp_kernel_4_2_2, GPUJPEG_YCBCR_BT601_256LVLS);
        } else {
            assert(false);
        }
    }
    // YCbCr color space
    else if ( coder->param_image.color_space == GPUJPEG_YCBCR_BT709 ) {
        if ( coder->param_image.sampling_factor == GPUJPEG_4_4_4 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_raw_to_comp_kernel_4_4_4, GPUJPEG_YCBCR_BT709);
        } else if ( coder->param_image.sampling_factor == GPUJPEG_4_2_2 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_raw_to_comp_kernel_4_2_2, GPUJPEG_YCBCR_BT709);
        } else {
            assert(false);
        }
    }
    // YUV color space
    else if ( coder->param_image.color_space == GPUJPEG_YUV ) {
        if ( coder->param_image.sampling_factor == GPUJPEG_4_4_4 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_raw_to_comp_kernel_4_4_4, GPUJPEG_YUV);
        } else if ( coder->param_image.sampling_factor == GPUJPEG_4_2_2 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_raw_to_comp_kernel_4_2_2, GPUJPEG_YUV);
        } else {
            assert(false);
        }
    }
    // Unknown color space
    else {
        assert(false);
    }
    
#undef RETURN_KERNEL_IF
#undef RETURN_KERNEL

    return NULL;
}

/** Documented at declaration */
int
gpujpeg_preprocessor_encoder_init(struct gpujpeg_coder* coder)
{
    if ( coder->param_image.comp_count == 1 ) {
        return 0;
    }

    assert(coder->param_image.comp_count == 3);

    if ( coder->param.color_space_internal == GPUJPEG_NONE ) {
        coder->preprocessor = (void*)gpujpeg_preprocessor_select_encode_kernel<GPUJPEG_NONE>(coder);
    } else if ( coder->param.color_space_internal == GPUJPEG_RGB ) {
        coder->preprocessor = (void*)gpujpeg_preprocessor_select_encode_kernel<GPUJPEG_RGB>(coder);
    } else if ( coder->param.color_space_internal == GPUJPEG_YCBCR_BT601_256LVLS ) {
        coder->preprocessor = (void*)gpujpeg_preprocessor_select_encode_kernel<GPUJPEG_YCBCR_BT601_256LVLS>(coder);
    } else {
        assert(false);
    }
    if ( coder->preprocessor == NULL )
        return -1;
    return 0;
}

/** Documented at declaration */
int
gpujpeg_preprocessor_encode(struct gpujpeg_coder* coder)
{    
    if ( coder->param_image.comp_count == 1 ) {
        hipMemcpy(coder->d_data, coder->d_data_raw, coder->data_raw_size * sizeof(uint8_t), hipMemcpyDeviceToDevice);
        return 0;
    }
    assert(coder->param_image.comp_count == 3);

    hipMemset(coder->d_data, 0, coder->data_size * sizeof(uint8_t));
    
    // Select kernel
    gpujpeg_preprocessor_encode_kernel kernel = (gpujpeg_preprocessor_encode_kernel)coder->preprocessor;
    assert(kernel != NULL);
         
    int image_width = coder->param_image.width;
    int image_height = coder->param_image.height;
    
    // When loading 4:2:2 data of odd width, the data in fact has even width, so round it
    // (at least imagemagick convert tool generates data stream in this way)
    if ( coder->param_image.sampling_factor == GPUJPEG_4_2_2 )
        image_width = (coder->param_image.width + 1) & ~1;
        
    // Prepare unit size
    assert(coder->param_image.sampling_factor == GPUJPEG_4_4_4 || coder->param_image.sampling_factor == GPUJPEG_4_2_2);
    int unitSize = coder->param_image.sampling_factor == GPUJPEG_4_4_4 ? 3 : 2;
    
    // Prepare kernel
    int alignedSize = gpujpeg_div_and_round_up(image_width * image_height, RGB_8BIT_THREADS) * RGB_8BIT_THREADS * unitSize;
    dim3 threads (RGB_8BIT_THREADS);
    dim3 grid (alignedSize / (RGB_8BIT_THREADS * unitSize));
    assert(alignedSize % (RGB_8BIT_THREADS * unitSize) == 0);
    while ( grid.x > GPUJPEG_CUDA_MAXIMUM_GRID_SIZE ) {
        grid.y *= 2;
        grid.x = gpujpeg_div_and_round_up(grid.x, 2);
    }
    
    // Decompose input image width for faster division using multiply-high and right shift
    uint32_t width_div_mul, width_div_shift;
    gpujpeg_const_div_prepare(image_width, width_div_mul, width_div_shift);
    
    // Run kernel
    struct gpujpeg_preprocessor_data data;
    for ( int comp = 0; comp < 3; comp++ ) {
        assert(coder->sampling_factor.horizontal % coder->component[comp].sampling_factor.horizontal == 0);
        assert(coder->sampling_factor.vertical % coder->component[comp].sampling_factor.vertical == 0);
        data.comp[comp].d_data = coder->component[comp].d_data;
        data.comp[comp].sampling_factor.horizontal = coder->sampling_factor.horizontal / coder->component[comp].sampling_factor.horizontal;
        data.comp[comp].sampling_factor.vertical = coder->sampling_factor.vertical / coder->component[comp].sampling_factor.vertical;
        data.comp[comp].data_width = coder->component[comp].data_width;
    }
    kernel<<<grid, threads>>>(
        data,
        coder->d_data_raw,
        coder->d_data_raw + coder->data_raw_size,
        image_width,
        image_height,
        width_div_mul,
        width_div_shift
    );
    hipDeviceSynchronize();
    gpujpeg_cuda_check_error("Preprocessor encoding failed", return -1);
        
    return 0;
}

/**
 * Store value to component data buffer in specified position by buffer size and subsampling
 * 
 * @param value
 * @param position_x
 * @param position_y
 * @param comp
 */
template<
    uint8_t s_samp_factor_h = GPUJPEG_DYNAMIC,
    uint8_t s_samp_factor_v = GPUJPEG_DYNAMIC
>
struct gpujpeg_preprocessor_comp_to_raw_load
{
    static __device__ void
    perform(uint8_t & value, int position_x, int position_y, struct gpujpeg_preprocessor_data_component & comp)
    {
        uint8_t samp_factor_h = s_samp_factor_h;
        if ( samp_factor_h == GPUJPEG_DYNAMIC ) {
            samp_factor_h = comp.sampling_factor.horizontal;
        }
        uint8_t samp_factor_v = s_samp_factor_v;
        if ( samp_factor_v == GPUJPEG_DYNAMIC ) {
            samp_factor_v = comp.sampling_factor.vertical;
        }
        
        position_x = position_x / samp_factor_h;
        position_y = position_y / samp_factor_v;
        
        int data_position = position_y * comp.data_width + position_x;
        value = comp.d_data[data_position];
    }
};
template<>
struct gpujpeg_preprocessor_comp_to_raw_load<1, 1>
{
    static __device__ void
    perform(uint8_t & value, int position_x, int position_y, struct gpujpeg_preprocessor_data_component & comp)
    {
        int data_position = position_y * comp.data_width + position_x;
        value = comp.d_data[data_position];
    }
};

/**
 * Kernel - Copy three separated component buffers into target image data
 *
 * @param d_c1  First component buffer
 * @param d_c2  Second component buffer
 * @param d_c3  Third component buffer
 * @param d_target  Image target data
 * @param pixel_count  Number of pixels to copy
 * @return void
 */
typedef void (*gpujpeg_preprocessor_decode_kernel)(struct gpujpeg_preprocessor_data data, uint8_t* d_data_raw, int image_width, int image_height);

/** Specialization [sampling factor is 4:4:4] */
template<
    enum gpujpeg_color_space color_space_internal,
    enum gpujpeg_color_space color_space,
    uint8_t s_comp1_samp_factor_h, uint8_t s_comp1_samp_factor_v,
    uint8_t s_comp2_samp_factor_h, uint8_t s_comp2_samp_factor_v,
    uint8_t s_comp3_samp_factor_h, uint8_t s_comp3_samp_factor_v
>
__global__ void
gpujpeg_preprocessor_comp_to_raw_kernel_4_4_4(struct gpujpeg_preprocessor_data data, uint8_t* d_data_raw, int image_width, int image_height)
{
    int x  = threadIdx.x;
    int gX = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
    int image_position = gX + x;
    if ( image_position >= (image_width * image_height) )
        return;
    int image_position_x = image_position % image_width;
    int image_position_y = image_position / image_width;
        
    // Load
    uint8_t r1;
    uint8_t r2;
    uint8_t r3;
    gpujpeg_preprocessor_comp_to_raw_load<s_comp1_samp_factor_h, s_comp1_samp_factor_v>::perform(r1, image_position_x, image_position_y, data.comp[0]);
    gpujpeg_preprocessor_comp_to_raw_load<s_comp2_samp_factor_h, s_comp2_samp_factor_v>::perform(r2, image_position_x, image_position_y, data.comp[1]);
    gpujpeg_preprocessor_comp_to_raw_load<s_comp3_samp_factor_h, s_comp3_samp_factor_v>::perform(r3, image_position_x, image_position_y, data.comp[2]);
    
    // Color transform
    gpujpeg_color_transform<color_space_internal, color_space>::perform(r1, r2, r3);
    
    // Store Order
    gpujpeg_color_order<color_space>::perform_store(r1, r2, r3);

    // Save
    image_position = image_position * 3;
    d_data_raw[image_position + 0] = r1;
    d_data_raw[image_position + 1] = r2;
    d_data_raw[image_position + 2] = r3;
}

/** Specialization [sampling factor is 4:2:2] */
template<
    enum gpujpeg_color_space color_space_internal,
    enum gpujpeg_color_space color_space,
    uint8_t s_comp1_samp_factor_h, uint8_t s_comp1_samp_factor_v,
    uint8_t s_comp2_samp_factor_h, uint8_t s_comp2_samp_factor_v,
    uint8_t s_comp3_samp_factor_h, uint8_t s_comp3_samp_factor_v
>
__global__ void
gpujpeg_preprocessor_comp_to_raw_kernel_4_2_2(struct gpujpeg_preprocessor_data data, uint8_t* d_data_raw, int image_width, int image_height)
{
    int x  = threadIdx.x;
    int gX = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
    int image_position = gX + x;
    if ( image_position >= (image_width * image_height) )
        return;
    int image_position_x = image_position % image_width;
    int image_position_y = image_position / image_width;
        
    // Load
    uint8_t r1;
    uint8_t r2;
    uint8_t r3;
    gpujpeg_preprocessor_comp_to_raw_load<s_comp1_samp_factor_h, s_comp1_samp_factor_v>::perform(r1, image_position_x, image_position_y, data.comp[0]);
    gpujpeg_preprocessor_comp_to_raw_load<s_comp2_samp_factor_h, s_comp2_samp_factor_v>::perform(r2, image_position_x, image_position_y, data.comp[1]);
    gpujpeg_preprocessor_comp_to_raw_load<s_comp3_samp_factor_h, s_comp3_samp_factor_v>::perform(r3, image_position_x, image_position_y, data.comp[2]);
    
    // Color transform
    gpujpeg_color_transform<color_space_internal, color_space>::perform(r1, r2, r3);
    
    // Store Order
    gpujpeg_color_order<color_space>::perform_store(r1, r2, r3);

    // Save
    image_position = image_position * 2;
    d_data_raw[image_position + 1] = r2;
    if ( (image_position_x % 2) == 0 )
        d_data_raw[image_position + 0] = r1;
    else
        d_data_raw[image_position + 0] = r3;
}

/**
 * Select preprocessor decode kernel
 * 
 * @param decoder
 * @return kernel
 */
template<enum gpujpeg_color_space color_space_internal>
gpujpeg_preprocessor_decode_kernel
gpujpeg_preprocessor_select_decode_kernel(struct gpujpeg_coder* coder)
{
    gpujpeg_preprocessor_sampling_factor_t sampling_factor = gpujpeg_preprocessor_make_sampling_factor(
        coder->sampling_factor.horizontal / coder->component[0].sampling_factor.horizontal,
        coder->sampling_factor.vertical / coder->component[0].sampling_factor.vertical,
        coder->sampling_factor.horizontal / coder->component[1].sampling_factor.horizontal,
        coder->sampling_factor.vertical / coder->component[1].sampling_factor.vertical,
        coder->sampling_factor.horizontal / coder->component[2].sampling_factor.horizontal,
        coder->sampling_factor.vertical / coder->component[2].sampling_factor.vertical
    );
    
#define RETURN_KERNEL_IF(KERNEL, COLOR, P1, P2, P3, P4, P5, P6) \
    if ( sampling_factor == gpujpeg_preprocessor_make_sampling_factor(P1, P2, P3, P4, P5, P6) ) { \
        int max_h = max(P1, max(P3, P5)); \
        int max_v = max(P2, max(P4, P6)); \
        if ( coder->param.verbose ) { \
            printf("Using faster kernel for postprocessor (precompiled %dx%d, %dx%d, %dx%d).\n", max_h / P1, max_v / P2, max_h / P3, max_v / P4, max_h / P5, max_v / P6); \
        } \
        return &KERNEL<color_space_internal, COLOR, P1, P2, P3, P4, P5, P6>; \
    } 
#define RETURN_KERNEL(KERNEL, COLOR) \
    RETURN_KERNEL_IF(KERNEL, COLOR, 1, 1, 1, 1, 1, 1) \
    else RETURN_KERNEL_IF(KERNEL, COLOR, 1, 1, 2, 2, 2, 2) \
    else RETURN_KERNEL_IF(KERNEL, COLOR, 1, 1, 1, 2, 1, 2) \
    else RETURN_KERNEL_IF(KERNEL, COLOR, 1, 1, 2, 1, 2, 1) \
    else RETURN_KERNEL_IF(KERNEL, COLOR, 1, 1, 4, 4, 4, 4) \
    else { \
        if ( coder->param.verbose ) { \
            printf("Using slower kernel for postprocessor (dynamic %dx%d, %dx%d, %dx%d).\n", coder->component[0].sampling_factor.horizontal, coder->component[0].sampling_factor.vertical, coder->component[1].sampling_factor.horizontal, coder->component[1].sampling_factor.vertical, coder->component[2].sampling_factor.horizontal, coder->component[2].sampling_factor.vertical); \
        } \
        return &KERNEL<color_space_internal, COLOR, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC, GPUJPEG_DYNAMIC>; \
    } \
    
    // None color space
    if ( coder->param_image.color_space == GPUJPEG_NONE ) {
        if ( coder->param_image.sampling_factor == GPUJPEG_4_4_4 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_comp_to_raw_kernel_4_4_4, GPUJPEG_NONE)
        } else if ( coder->param_image.sampling_factor == GPUJPEG_4_2_2 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_comp_to_raw_kernel_4_2_2, GPUJPEG_NONE)
        } else {
            assert(false);
        }
    } 
    // RGB color space
    else if ( coder->param_image.color_space == GPUJPEG_RGB ) {
        if ( coder->param_image.sampling_factor == GPUJPEG_4_4_4 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_comp_to_raw_kernel_4_4_4, GPUJPEG_RGB)
        } else if ( coder->param_image.sampling_factor == GPUJPEG_4_2_2 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_comp_to_raw_kernel_4_2_2, GPUJPEG_RGB)
        } else {
            assert(false);
        }
    }
    // YCbCr color space
    else if ( coder->param_image.color_space == GPUJPEG_YCBCR_BT601 ) {
        if ( coder->param_image.sampling_factor == GPUJPEG_4_4_4 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_comp_to_raw_kernel_4_4_4, GPUJPEG_YCBCR_BT601)
        } else if ( coder->param_image.sampling_factor == GPUJPEG_4_2_2 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_comp_to_raw_kernel_4_2_2, GPUJPEG_YCBCR_BT601)
        } else {
            assert(false);
        }
    }
    // YCbCr color space
    else if ( coder->param_image.color_space == GPUJPEG_YCBCR_BT601_256LVLS ) {
        if ( coder->param_image.sampling_factor == GPUJPEG_4_4_4 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_comp_to_raw_kernel_4_4_4, GPUJPEG_YCBCR_BT601_256LVLS)
        } else if ( coder->param_image.sampling_factor == GPUJPEG_4_2_2 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_comp_to_raw_kernel_4_2_2, GPUJPEG_YCBCR_BT601_256LVLS)
        } else {
            assert(false);
        }
    }
    // YCbCr color space
    else if ( coder->param_image.color_space == GPUJPEG_YCBCR_BT709 ) {
        if ( coder->param_image.sampling_factor == GPUJPEG_4_4_4 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_comp_to_raw_kernel_4_4_4, GPUJPEG_YCBCR_BT709)
        } else if ( coder->param_image.sampling_factor == GPUJPEG_4_2_2 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_comp_to_raw_kernel_4_2_2, GPUJPEG_YCBCR_BT709)
        } else {
            assert(false);
        }
    }
    // YUV color space
    else if ( coder->param_image.color_space == GPUJPEG_YUV ) {
        if ( coder->param_image.sampling_factor == GPUJPEG_4_4_4 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_comp_to_raw_kernel_4_4_4, GPUJPEG_YUV)
        } else if ( coder->param_image.sampling_factor == GPUJPEG_4_2_2 ) {
            RETURN_KERNEL(gpujpeg_preprocessor_comp_to_raw_kernel_4_2_2, GPUJPEG_YUV)
        } else {
            assert(false);
        }
    }
    // Unknown color space
    else {
        assert(false);
    }
    
#undef RETURN_KERNEL_IF
#undef RETURN_KERNEL
    
    return NULL;
}

/** Documented at declaration */
int
gpujpeg_preprocessor_decoder_init(struct gpujpeg_coder* coder)
{
    if ( coder->param_image.comp_count == 1 ) {
        return 0;
    }

    assert(coder->param_image.comp_count == 3);

    if ( coder->param.color_space_internal == GPUJPEG_NONE ) {
        coder->preprocessor = (void*)gpujpeg_preprocessor_select_decode_kernel<GPUJPEG_NONE>(coder);
    } else if ( coder->param.color_space_internal == GPUJPEG_RGB ) {
        coder->preprocessor = (void*)gpujpeg_preprocessor_select_decode_kernel<GPUJPEG_RGB>(coder);
    } else if ( coder->param.color_space_internal == GPUJPEG_YCBCR_BT601_256LVLS ) {
        coder->preprocessor = (void*)gpujpeg_preprocessor_select_decode_kernel<GPUJPEG_YCBCR_BT601_256LVLS>(coder);
    } else {
        assert(false);
    }
    if ( coder->preprocessor == NULL )
        return -1;
    return 0;
}

/** Documented at declaration */
int
gpujpeg_preprocessor_decode(struct gpujpeg_coder* coder)
{
    if ( coder->param_image.comp_count == 1 ) {
        hipMemcpy(coder->d_data_raw, coder->d_data, coder->data_raw_size * sizeof(uint8_t), hipMemcpyDeviceToDevice);
        return 0;
    }
    assert(coder->param_image.comp_count == 3);

    hipMemset(coder->d_data_raw, 0, coder->data_raw_size * sizeof(uint8_t));
    
    // Select kernel
    gpujpeg_preprocessor_decode_kernel kernel = (gpujpeg_preprocessor_decode_kernel)coder->preprocessor;
    assert(kernel != NULL);
    
    int image_width = coder->param_image.width;
    int image_height = coder->param_image.height;
    
    // When saving 4:2:2 data of odd width, the data should have even width, so round it
    if ( coder->param_image.sampling_factor == GPUJPEG_4_2_2 )
        image_width = gpujpeg_div_and_round_up(coder->param_image.width, 2) * 2;
        
    // Prepare unit size
    assert(coder->param_image.sampling_factor == GPUJPEG_4_4_4 || coder->param_image.sampling_factor == GPUJPEG_4_2_2);
    int unitSize = coder->param_image.sampling_factor == GPUJPEG_4_4_4 ? 3 : 2;
    
    // Prepare kernel
    int alignedSize = gpujpeg_div_and_round_up(image_width * image_height, RGB_8BIT_THREADS) * RGB_8BIT_THREADS * unitSize;
    dim3 threads (RGB_8BIT_THREADS);
    dim3 grid (alignedSize / (RGB_8BIT_THREADS * unitSize));
    assert(alignedSize % (RGB_8BIT_THREADS * unitSize) == 0);
    if ( grid.x > GPUJPEG_CUDA_MAXIMUM_GRID_SIZE ) {
        grid.y = gpujpeg_div_and_round_up(grid.x, GPUJPEG_CUDA_MAXIMUM_GRID_SIZE);
        grid.x = GPUJPEG_CUDA_MAXIMUM_GRID_SIZE;
    }

    // Run kernel
    struct gpujpeg_preprocessor_data data;
    for ( int comp = 0; comp < 3; comp++ ) {
        assert(coder->sampling_factor.horizontal % coder->component[comp].sampling_factor.horizontal == 0);
        assert(coder->sampling_factor.vertical % coder->component[comp].sampling_factor.vertical == 0);
        data.comp[comp].d_data = coder->component[comp].d_data;
        data.comp[comp].sampling_factor.horizontal = coder->sampling_factor.horizontal / coder->component[comp].sampling_factor.horizontal;
        data.comp[comp].sampling_factor.vertical = coder->sampling_factor.vertical / coder->component[comp].sampling_factor.vertical;
        data.comp[comp].data_width = coder->component[comp].data_width;
    }
    kernel<<<grid, threads>>>(
        data,
        coder->d_data_raw, 
        image_width,
        image_height
    );
    hipDeviceSynchronize();
    gpujpeg_cuda_check_error("Preprocessor encoding failed", return -1);
    
    return 0;
}
